#include "hip/hip_runtime.h"
#include "searcher_cuda.h"

#include <cuda/atomic>

#include <iostream>
#include <format>
#include <unistd.h>
#include <cstdio>

#define MAX_FCFS 16384
#define MAX_SOLUTIONS (1ull << 24)
#define fcf_threads 64

#define C(ans) { chk_impl((ans), __FILE__, __LINE__); }

static inline void chk_impl(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << std::format("CUDA: {} @ {}:{}\n", hipGetErrorString(code), file, line);
    }
}

__device__ static tt_t fcf[MAX_FCFS];
__device__ static size_t shps;
__device__ static size_t fcfs;

void fcf_cache(size_t num_shapes) {
    C(hipMemcpyToSymbol(HIP_SYMBOL(fcf), fast_canonical_form, fast_canonical_forms * sizeof(tt_t)));
    C(hipMemcpyToSymbol(HIP_SYMBOL(shps), &num_shapes, sizeof(size_t)));
    C(hipMemcpyToSymbol(HIP_SYMBOL(fcfs), &fast_canonical_forms, sizeof(size_t)));
    C(hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, ~0ull));
    size_t drplc;
    C(hipDeviceGetLimit(&drplc, cudaLimitDevRuntimePendingLaunchCount));
    std::cout << std::format("DRPLC = {}\n", drplc);
}

template <unsigned D, bool DP = true> // 0 ~ 27
__global__
void searcher_impl(uint64_t empty_area,
        CudaSearcher::R *solutions, uint32_t *n_solutions, uint32_t *n_pending,
        uint32_t ex0, uint32_t ex1, uint32_t ex2, uint32_t ex3,
        uint32_t ex4, uint32_t ex5, uint32_t ex6) {
    uint64_t nms{}, nmm{}, covering{}, shape{};
    uint32_t nmx{};
    uint8_t nm{};

    auto idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= fcfs) goto fin;
    covering = empty_area & -empty_area; // Shape<8>::front_shape();
    shape = fcf[idx].shape;
    nm = fcf[idx].nm;
    if (!(shape & covering)) [[likely]] goto fin;
    if (shape & ~empty_area) [[likely]] goto fin;
    nmx = __byte_perm(nm, 0, 0); // nm nm nm nm
    if constexpr (D) {
        if constexpr (D >  0) if (__vcmpeq4(nmx, ex0)) [[unlikely]] goto fin;
        if constexpr (D >  4) if (__vcmpeq4(nmx, ex1)) [[unlikely]] goto fin;
        if constexpr (D >  8) if (__vcmpeq4(nmx, ex2)) [[unlikely]] goto fin;
        if constexpr (D > 12) if (__vcmpeq4(nmx, ex3)) [[unlikely]] goto fin;
        if constexpr (D > 16) if (__vcmpeq4(nmx, ex4)) [[unlikely]] goto fin;
        if constexpr (D > 20) if (__vcmpeq4(nmx, ex5)) [[unlikely]] goto fin;
        if constexpr (D > 24) if (__vcmpeq4(nmx, ex6)) [[unlikely]] goto fin;
    }
    nms = static_cast<uint64_t>(nm) << (D % 4) * 8;
    nmm = static_cast<uint64_t>(0xff) << (D % 4) * 8;
         if constexpr (D <  4) ex0 = ((ex0 & ~nmm) | nms);
    else if constexpr (D <  8) ex1 = ((ex1 & ~nmm) | nms);
    else if constexpr (D < 12) ex2 = ((ex2 & ~nmm) | nms);
    else if constexpr (D < 16) ex3 = ((ex3 & ~nmm) | nms);
    else if constexpr (D < 20) ex4 = ((ex4 & ~nmm) | nms);
    else if constexpr (D < 24) ex5 = ((ex5 & ~nmm) | nms);
    else if constexpr (D < 28) ex6 = ((ex6 & ~nmm) | nms);
    if (!(empty_area & ~shape)) {
        auto pos = atomicAdd(n_solutions, 1);
        auto &ret = solutions[pos % MAX_SOLUTIONS];
        ret.empty_area = 0u;
        ret.ex[0] = (D >=  0) ? ex0 : ~0u;
        ret.ex[1] = (D >=  4) ? ex1 : ~0u;
        ret.ex[2] = (D >=  8) ? ex2 : ~0u;
        ret.ex[3] = (D >= 12) ? ex3 : ~0u;
        ret.ex[4] = (D >= 16) ? ex4 : ~0u;
        ret.ex[5] = (D >= 20) ? ex5 : ~0u;
        ret.ex[6] = (D >= 24) ? ex6 : ~0u;
        ret.d = D;
        atomicAdd(n_solutions + 1, 1);
        goto fin;
    }
    if constexpr (D < 28) {
        auto err = cudaErrorLaunchPendingCountExceeded;
        auto fcf_blocks = (fcfs + fcf_threads - 1) / fcf_threads;
        if constexpr (DP) {
            atomicAdd(n_pending, fcf_blocks * fcf_threads);
            searcher_impl<D + 1><<<fcf_blocks, fcf_threads, 0, cudaStreamFireAndForget>>>(
                    empty_area & ~shape, solutions, n_solutions, n_pending,
                    ex0, ex1, ex2, ex3, ex4, ex5, ex6);
            err = hipPeekAtLastError();
            if (err == hipSuccess)
                goto fin;
        }
        auto pos = atomicAdd(n_solutions, 1);
        auto &ret = solutions[pos % MAX_SOLUTIONS];
        ret.ex[0] = (D >=  0) ? ex0 : ~0u;
        ret.ex[1] = (D >=  4) ? ex1 : ~0u;
        ret.ex[2] = (D >=  8) ? ex2 : ~0u;
        ret.ex[3] = (D >= 12) ? ex3 : ~0u;
        ret.ex[4] = (D >= 16) ? ex4 : ~0u;
        ret.ex[5] = (D >= 20) ? ex5 : ~0u;
        ret.ex[6] = (D >= 24) ? ex6 : ~0u;
        if (err == cudaErrorLaunchPendingCountExceeded) {
            ret.empty_area = empty_area & ~shape;
            ret.d = D + 1;
        } else {
            ret.empty_area = err;
            ret.d = 0x5555aaaa;
        }
        atomicAdd(n_solutions + 1, 1);
        if constexpr (DP)
            atomicSub(n_pending, fcf_blocks * fcf_threads);
        goto fin;
    }
fin:
    atomicSub(n_pending, 1);
}

CudaSearcher::CudaSearcher(size_t num_shapes)
    : solutions{}, n_solutions{}, n_solution_processed{}, n_kernel_invoked{}, n_pending{} {
    C(hipMallocManaged(&solutions, MAX_SOLUTIONS * sizeof(R)));
    C(hipMallocManaged(&n_solutions, 2 * sizeof(n_solutions)));
    C(hipMallocManaged(&n_pending, sizeof(n_pending)));
}

CudaSearcher::~CudaSearcher() {
    C(hipFree(solutions));
    C(hipFree(const_cast<uint32_t *>(n_solutions)));
    C(hipFree(const_cast<uint32_t *>(n_pending)));
}

void CudaSearcher::start_search(uint64_t empty_area) {
    n_solutions[0] = 0;
    n_solutions[1] = 0;
    *n_pending = 0;
    invoke_kernel(R{ empty_area, 0u, { ~0u, ~0u, ~0u, ~0u, ~0u, ~0u, ~0u } });
}

void CudaSearcher::invoke_kernel(const R &args) {
    // std::cerr << std::format("~{}th invoking {} kernel @ ea={}/{:016x} ex={:08x}{:08x}{:08x}{:08x}{:08x}{:08x}{:08x}",
    //         n_kernel_invoked++, args.d,
    //         std::popcount(args.empty_area), args.empty_area,
    //         args.ex[0], args.ex[1], args.ex[2], args.ex[3],
    //         args.ex[4], args.ex[5], args.ex[6]);
    for (auto wait = 1u; ; wait = wait >= 1000000u ? 1000000u : 2 * wait) {
        auto fcf_blocks = (fast_canonical_forms + fcf_threads - 1) / fcf_threads;
        auto err = hipSuccess;
        if (false)
            ;
#define INV(D) \
        else if (args.d == D) { \
            searcher_impl<D><<<fcf_blocks, fcf_threads>>>( \
                args.empty_area, solutions, \
                const_cast<uint32_t *>(n_solutions), \
                const_cast<uint32_t *>(n_pending), \
                args.ex[0], args.ex[1], args.ex[2], args.ex[3], \
                args.ex[4], args.ex[5], args.ex[6]); \
            err = hipPeekAtLastError(); }
        INV( 0) INV( 1) INV( 2) INV( 3)
        INV( 4) INV( 5) INV( 6) INV( 7)
        INV( 8) INV( 9) INV(10) INV(11)
        INV(12) INV(13) INV(14) INV(15)
        INV(16) INV(17) INV(18) INV(19)
        INV(20) INV(21) INV(22) INV(23)
        INV(24) INV(25) INV(26) INV(27)
#undef INV
        if (err == cudaErrorLaunchPendingCountExceeded) {
            std::cerr<< '.';
            continue;
        }
        C(err);
        // no need to worry about ordering - we are the host thread
        cuda::atomic_ref pd{ *const_cast<uint32_t *>(n_pending) };
        pd += fcf_blocks * fcf_threads;
        return;
    }
}

const unsigned char *CudaSearcher::next() {
    auto flag = false;
    auto old_val = 0u;
    // auto o = *n_pending;
    for (auto wait = 1u; ; wait = wait >= 1000000u ? 1000000u : 2 * wait) {
        auto curr = n_solutions[1];
    again:
        if (curr > n_solution_processed) {
            auto &ret = solutions[n_solution_processed++ % MAX_SOLUTIONS];
            if (ret.d == 0x5555aaaau) {
                auto err = static_cast<hipError_t>(ret.empty_area);
                throw std::runtime_error{ std::format("{} at #{}: {}",
                        hipGetErrorName(err), curr, hipGetErrorString(err)) };
            } else if (ret.empty_area) {
                invoke_kernel(ret);
                flag = false;
                goto again;
            }
            return reinterpret_cast<unsigned char *>(ret.ex);
        }
        if (flag)
            return nullptr;
        auto val = *n_pending;
        if (!val) {
            flag = true;
            continue;
        }
        if (wait >= 10000000u && val != old_val) {
            std::cerr << std::format("n_pending = {}\n", val);
            old_val = val;
        }
        usleep(wait);
    }
}

void show_devices() {
  int nDevices;
  C(hipGetDeviceCount(&nDevices));

  printf("Number of devices: %d\n", nDevices);

  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    C(hipGetDeviceProperties(&prop, i));
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Capability: %d.%d\n", prop.major, prop.minor);
    printf("  MP: %d\n", prop.multiProcessorCount);
    printf("  Memory Clock Rate (MHz): %d\n", prop.memoryClockRate/1024);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %.1f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Warp-size: %d\n", prop.warpSize);
    printf("  Threads/block: %d\n", prop.maxThreadsPerBlock);
    printf("  Threads/mp: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  Async engines: %d\n", prop.asyncEngineCount);
    printf("  32-bit Registers per block: %d\n", prop.regsPerBlock);
    printf("  32-bit Registers per mp: %d\n", prop.regsPerMultiprocessor);
    printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
    printf("  Concurrent computation/communication: %s\n",prop.deviceOverlap ? "yes" : "no");
    printf("  Sparse: %s\n",prop.sparseCudaArraySupported ? "yes" : "no");
    printf("  Managed mem: %s\n",prop.managedMemory ? "yes" : "no");
    printf("  Deferred: %s\n",prop.deferredMappingCudaArraySupported ? "yes" : "no");
    printf("  Map host mem: %s\n",prop.canMapHostMemory ? "yes" : "no");
    printf("  Unified addr: %s\n",prop.unifiedAddressing ? "yes" : "no");
    printf("  Unified fp: %s\n",prop.unifiedFunctionPointers ? "yes" : "no");
    printf("  Concurrent managed access: %s\n",prop.concurrentManagedAccess ? "yes" : "no");
    printf("  PMA: %s\n",prop.pageableMemoryAccess ? "yes" : "no");
    printf("  ECC: %s\n",prop.ECCEnabled ? "yes" : "no");
    printf("  Cooperative launch: %s\n",prop.cooperativeLaunch ? "yes" : "no");
    printf("  DMMA from host: %s\n",prop.directManagedMemAccessFromHost ? "yes" : "no");
    printf("  L2 Cache Size (KiB): %d\n",prop.l2CacheSize / 1024);
    printf("  Shared mem per block (KiB): %lu\n",prop.sharedMemPerBlock / 1024);
    printf("  Shared mem per mp (KiB): %lu\n",prop.sharedMemPerMultiprocessor / 1024);
    printf("  Const mem (B): %lu\n",prop.totalConstMem / 1024);
    printf("  Global mem (MiB): %lf\n",prop.totalGlobalMem / 1024.0 / 1024);
  }
}
