#include "hip/hip_runtime.h"
#include "searcher_cuda.h"
#include "growable.cuh"
#include "sn.cuh"

#include <hip/hip_runtime.h>
#include <cuda/atomic>
#include <cstring>
#include <memory>
#include <iostream>
#include <format>
#include <unistd.h>
#include <cstdio>

#define MAX_SOLUTIONS (1ull << 24)

/**
 * 128 resident grids / device (Concurrent Kernel Execution)
 * 2147483647*65535*65535 blocks / grid
 * 1024*1024*64 <= 1024 threads / block
 * 32 threads / warp
 * 16 blocks / SM
 * 48 threads / warp
 * 1536 threads / SM
 * 65536 regs / SM
 * 255 regs / threads
 * 64KiB constant memory (8KiB cache)
 */

void chk_impl(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        throw std::runtime_error{
            std::format("CUDA: {}: {} @ {}:{}\n",
                    hipGetErrorName(code), hipGetErrorString(code),
                    file, line) };
    }
}

void chk_impl(hipError_t code, const char *file, int line) {
    const char *pn = "???", *ps = "???";
    hipDrvGetErrorName(code, &pn);
    hipDrvGetErrorString(code, &ps);
    if (code != hipSuccess) {
        throw std::runtime_error{
            std::format("CUDA Driver: {}: {} @ {}:{}\n", pn, ps, file, line) };
    }
}

static int n_devices;
static const frow_info_t *h_frowInfoL, *h_frowInfoR;
static frow_t *d_frowDataL[128][16], *d_frowDataR[128][16];

void frow_cache(const frow_info_t *fiL, const frow_info_t *fiR) {
    C(hipGetDeviceCount(&n_devices));
    n_devices = min(n_devices, 128);
    std::cout << std::format("n_devices = {}\n", n_devices);
    if (!n_devices)
        throw std::runtime_error{ "no CUDA device" };

    h_frowInfoL = fiL;
    h_frowInfoR = fiR;
    for (auto d = 0; d < n_devices; d++) {
        C(hipSetDevice(d));
        for (auto i = 0; i < 16; i++) {
            C(hipMalloc(&d_frowDataL[d][i], fiL[i].sz[5] * sizeof(frow_t)));
            C(hipMalloc(&d_frowDataR[d][i], fiR[i].sz[5] * sizeof(frow_t)));
            C(hipMemcpyAsync(d_frowDataL[d][i], fiL[i].data,
                        fiL[i].sz[5] * sizeof(frow_t), hipMemcpyHostToDevice));
            C(hipMemcpyAsync(d_frowDataR[d][i], fiR[i].data,
                        fiR[i].sz[5] * sizeof(frow_t), hipMemcpyHostToDevice));
        }
        C(hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, ~0ull));
        size_t drplc;
        C(hipDeviceGetLimit(&drplc, cudaLimitDevRuntimePendingLaunchCount));
        std::cout << std::format("dev{}.DRPLC = {}\n", d, drplc);
    }
}

#define CYC_CHUNK (10ull * 1048576ull / sizeof(R))

__global__
void d_row_search(
        R *bins,
        unsigned long long *n_bins,
        const uint32_t *n_available_chunks,
        uint32_t *n_completed_chunks,
        const R *cfgs, uint64_t n_cfgs,
        const frow_t *f0L, uint32_t f0Lsz,
        const frow_t *f0R, uint32_t f0Rsz) {
    auto idx = threadIdx.x + static_cast<uint64_t>(blockIdx.x) * blockDim.x;
    if (idx >= n_cfgs * f0Lsz * f0Rsz) [[unlikely]] return;
    auto cfg = cfgs[idx / f0Rsz / f0Lsz];
    auto fL  = f0L [idx / f0Rsz % f0Lsz];
    auto fR  = f0R [idx % f0Rsz];
    if (fL.shape & ~cfg.empty_area) [[unlikely]] return;
    if (fR.shape & ~cfg.empty_area) [[unlikely]] return;
    if (fL.shape & fR.shape) [[unlikely]] return;
    d_push(cfg.nm_cnt, cfg.ex, fL.nm0123);
    d_push(cfg.nm_cnt, cfg.ex, fR.nm0123);
    d_sn(cfg.nm_cnt, cfg.ex);
    if (!d_uniq_chk(cfg.nm_cnt, cfg.ex)) [[unlikely]] return;
    cfg.empty_area &= ~fL.shape;
    cfg.empty_area &= ~fR.shape;
    __builtin_assume(!(cfg.empty_area & 0b11111111u));
    cfg.empty_area >>= 8;
    auto out = __nv_atomic_fetch_add(n_bins, 1,
            __NV_ATOMIC_ACQUIRE, __NV_THREAD_SCOPE_DEVICE);
    unsigned long long cap;
spin:
    cap = __nv_atomic_load_n(const_cast<uint32_t *>(n_available_chunks),
            __NV_ATOMIC_ACQUIRE, __NV_THREAD_SCOPE_SYSTEM) * CYC_CHUNK;
    if (out >= cap)
        goto spin;
    bins[out] = cfg; // slice
    if (out % CYC_CHUNK == 0) {
        __nv_atomic_fetch_add(n_completed_chunks, 1,
                __NV_ATOMIC_RELEASE, __NV_THREAD_SCOPE_SYSTEM);
    }
}

CudaSearcher::CudaSearcher(uint64_t empty_area)
    : solutions{}, grs{},
      height{ (std::bit_width(empty_area) + 8u - 1u) / 8u } {
    auto &r = solutions[empty_area & 0xffu];
    C(hipMallocManaged(&r.ptr, sizeof(R)));
    r.ptr[0] = R{ empty_area, { ~0u, ~0u, ~0u, ~0u }, 0 };
    r.len = 1;
}

CudaSearcher::~CudaSearcher() {
    free();
}

void CudaSearcher::free() {
    for (auto &r : solutions) {
        if (r.ptr)
            hipFree(r.ptr);
        r.ptr = nullptr;
        r.len = 0;
    }
}

std::pair<uint64_t, uint32_t> balance(uint64_t n) {
    if (n <= 32)
        return { 1, n };
    if (n <= 32 * 84 * 3)
        return { (n + 31) / 32, 32 };
    if (n <= 64 * 84 * 3)
        return { (n + 63) / 64, 64 };
    if (n <= 96 * 84 * 3)
        return { (n + 95) / 96, 96 };
    if (n <= 128 * 84 * 3)
        return { (n + 127) / 128, 128 };
    if (n <= 256 * 84 * 3)
        return { (n + 255) / 256, 256 };
    return { (n + 511) / 512, 512 };
}

#define VMEM_SZ ((1zu << 43) / sizeof(R))

struct Device {
    int dev;
    hipStream_t stream;

    std::unique_ptr<Growable<R>> gr;

    uint32_t *counters;
    uint32_t n_collected_chunks;

    R *bins; // __device__, but owned by Growable<R>
    unsigned long long *n_bins; // __device__, owned

    Device(Device &&other) = default;
    Device &operator=(Device &&other) = default;
    explicit Device(int d)
        : dev{ d }, stream{}, gr{},
          counters{}, n_collected_chunks{}, bins{}, n_bins{} {
        C(hipMallocManaged(&counters, 2 * sizeof(uint32_t)));

        C(hipSetDevice(d));
        C(hipStreamCreate(&stream));
        C(hipMallocAsync(&n_bins, sizeof(unsigned long long), stream));

        std::cout << std::format("dev#{}: map {}B of vmem, then fill it with {}B chunks\n",
                dev, display(VMEM_SZ), display(CYC_CHUNK * sizeof(R)));
        gr = std::make_unique<Growable<R>>(VMEM_SZ);
        auto k = 0;
        while (true) {
            if (auto p = gr->get(CYC_CHUNK * ++k); p)
                bins = p;
            else
                break;
        }
        k--;
        cuda::atomic_ref n_available_chunks{ counters[0] };
        n_available_chunks.fetch_add(k, cuda::memory_order_release);
        std::cout << std::format("dev#{}: {} * {}B = {}B of mem ({}) mapped\n",
                dev, k, display(CYC_CHUNK * sizeof(R)), display(k * CYC_CHUNK * sizeof(R)), k * CYC_CHUNK);
    }

    ~Device() {
        C(hipSetDevice(dev));
        C(hipFreeAsync(counters, stream));
        C(hipFreeAsync(n_bins, stream));
        C(hipStreamSynchronize(stream));
        C(hipStreamDestroy(stream));
    }

    [[nodiscard]] bool ready() const {
        auto res = hipStreamQuery(stream);
        switch (res) {
            case hipSuccess: return true;
            case hipErrorNotReady: return false;
            default: C(res); return false;
        }
    }

    void join() {
        std::cout << std::format("dev#{}: synchronize\n", dev);
        C(hipStreamSynchronize(stream));
    }

    void dispatch(unsigned pos, unsigned height, Rg<R> cfgs) {
        auto [ptr, len] = cfgs;
        auto szid = min(height - 1, 5);
        auto fanoutL = h_frowInfoL[(pos >> 0) & 0b1111u].sz[szid];
        auto fanoutR = h_frowInfoR[(pos >> 4) & 0b1111u].sz[szid];
        auto sz = len * fanoutL * fanoutR;
        auto d_f0L = d_frowDataL[dev][pos >> 0 & 0xfu];
        auto d_f0R = d_frowDataR[dev][pos >> 4 & 0xfu];
        auto [b, t] = balance(sz);
        std::cout << std::format("#{:08b}-dev#{}: <<<{}, {}>>> = {} * L{} * R{} => max {}B\n",
                pos, dev, b, t, len, fanoutL, fanoutR, display(sz * sizeof(R)));
        C(hipSetDevice(dev));
        C(hipStreamAttachMemAsync(stream, ptr, len * sizeof(R)));
        C(hipMemPrefetchAsync(ptr, len * sizeof(R), dev, stream));
        d_row_search<<<b, t, 0, stream>>>(bins, n_bins,
                &counters[0], &counters[1],
                ptr, len,
                d_f0L, fanoutL,
                d_f0R, fanoutR);
        C(hipFreeAsync(ptr, stream));
    }

    void collect(Sorter &sorter, bool force = false) {
        cuda::atomic_ref n_available_chunks{ counters[0] };
        cuda::atomic_ref n_completed_chunks{ counters[1] };
        auto completed = n_completed_chunks.load(cuda::memory_order_acquire);
        auto k = completed > n_collected_chunks ? completed - n_collected_chunks : 0u;
        if (k) {
            auto n = n_available_chunks.load(cuda::memory_order_relaxed);
            std::cout << std::format("dev#{}: {}/{} (+{}) chunks ({}B) recycled\n",
                    dev, completed, n, k, display(k * CYC_CHUNK * sizeof(R)));
        }
        for (auto i = 0u; i < k; i++) {
            gr->commit(CYC_CHUNK);
            gr->evict1();
            n_available_chunks.fetch_add(1, cuda::memory_order_acquire);
            completed++;
        }
        if (!force && !sorter.ready())
            return;
        std::cout << std::format("dev#{}: pushing {} entries to sorter ({}B)\n",
                dev, gr->get_load(), display(gr->get_load() * sizeof(R)));
        sorter.push(gr->remove_data());
    }
};

void CudaSearcher::search_GPU() {
    Sorter sorter{ *this };
    std::vector<Device> devs;
    for (auto i = 0; i < n_devices; i++)
        devs.emplace_back(i);

    for (unsigned ipos = 0u, dev = 0u; ipos <= 255u; ipos++) {
        devs[dev].dispatch(ipos, height, solutions[ipos]);
        dev = (dev + 1) % n_devices;
    }
    bool flag;
    do {
        flag = true;
        for (auto &dev : devs) {
            flag &= dev.ready();
            dev.collect(sorter);
        }
    } while (!flag);
    for (auto &dev : devs)
        dev.collect(sorter, true);
    devs.clear();
    sorter.join();
}

uint64_t CudaSearcher::next_size(unsigned pos) const {
    auto szid = min(height - 1, 5);
    return solutions[pos].len
        * h_frowInfoL[(pos >> 0) & 0b1111u].sz[szid]
        * h_frowInfoR[(pos >> 4) & 0b1111u].sz[szid];
}

Rg<R> CudaSearcher::write_solution(unsigned pos, size_t sz) {
    auto &r = solutions[pos];
    C(hipFree(r.ptr));
    r.ptr = nullptr, r.len = 0;
    C(hipMallocManaged(&r.ptr, sz * sizeof(R), hipMemAttachHost));
    r.len = sz;
    return r;
}

void show_devices() {
  int nDevices;
  C(hipGetDeviceCount(&nDevices));

  printf("Number of devices: %d\n", nDevices);

  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    C(hipGetDeviceProperties(&prop, i));
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Capability: %d.%d\n", prop.major, prop.minor);
    printf("  MP: %d\n", prop.multiProcessorCount);
    printf("  Memory Clock Rate (MHz): %d\n", prop.memoryClockRate/1024);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %.1f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Warp-size: %d\n", prop.warpSize);
    printf("  Threads/block: %d\n", prop.maxThreadsPerBlock);
    printf("  Threads/mp: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  Async engines: %d\n", prop.asyncEngineCount);
    printf("  32-bit Registers per block: %d\n", prop.regsPerBlock);
    printf("  32-bit Registers per mp: %d\n", prop.regsPerMultiprocessor);
    printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
    printf("  Concurrent computation/communication: %s\n",prop.deviceOverlap ? "yes" : "no");
    printf("  Sparse: %s\n",prop.sparseCudaArraySupported ? "yes" : "no");
    printf("  Managed mem: %s\n",prop.managedMemory ? "yes" : "no");
    printf("  Deferred: %s\n",prop.deferredMappingCudaArraySupported ? "yes" : "no");
    printf("  Map host mem: %s\n",prop.canMapHostMemory ? "yes" : "no");
    printf("  Unified addr: %s\n",prop.unifiedAddressing ? "yes" : "no");
    printf("  Unified fp: %s\n",prop.unifiedFunctionPointers ? "yes" : "no");
    printf("  Concurrent managed access: %s\n",prop.concurrentManagedAccess ? "yes" : "no");
    printf("  PMA: %s\n",prop.pageableMemoryAccess ? "yes" : "no");
    printf("  ECC: %s\n",prop.ECCEnabled ? "yes" : "no");
    printf("  Cooperative launch: %s\n",prop.cooperativeLaunch ? "yes" : "no");
    printf("  DMMA from host: %s\n",prop.directManagedMemAccessFromHost ? "yes" : "no");
    printf("  L2 Cache Size (KiB): %d\n",prop.l2CacheSize / 1024);
    printf("  Shared mem per block (KiB): %lu\n",prop.sharedMemPerBlock / 1024);
    printf("  Shared mem per mp (KiB): %lu\n",prop.sharedMemPerMultiprocessor / 1024);
    printf("  Const mem (B): %lu\n",prop.totalConstMem / 1024);
    printf("  Global mem (MiB): %lf\n",prop.totalGlobalMem / 1024.0 / 1024);
    int v;
    hipDeviceGetAttribute(&v, cudaDevAttrMemSyncDomainCount, i);
    printf("  Sync domain: %d\n",v);
    hipDeviceGetAttribute(&v, hipDeviceAttributeSingleToDoublePrecisionPerfRatio, i);
    printf("  float/double ratio: %d\n", v);
    hipDeviceGetAttribute(&v, (hipDeviceAttribute_t)102, i);
    printf("  VMM: %d\n", v);
  }
}
