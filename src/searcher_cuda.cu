#include "hip/hip_runtime.h"
#include "searcher_cuda.h"
#include "util.hpp"
#include "util.cuh"
#include "frow.h"
#include "sn.cuh"

#include <hip/hip_runtime.h>
#include <cuda/atomic>
#include <algorithm>
#include <cstring>
#include <memory>
#include <deque>
#include <iostream>
#include <format>
#include <unistd.h>
#include <cstdio>

/**
 * 128 resident grids / device (Concurrent Kernel Execution)
 * 2147483647*65535*65535 blocks / grid
 * 1024*1024*64 <= 1024 threads / block
 * 32 threads / warp
 * 16 blocks / SM
 * 48 threads / warp
 * 1536 threads / SM
 * 65536 regs / SM
 * 255 regs / threads
 * 64KiB constant memory (8KiB cache)
 */

#define CYC_CHUNK (32ull * 1048576ull / sizeof(RX))

template <unsigned H>
__global__
void d_row_search(
        // output ring buffer
        RX                 *ring_buffer, // __device__
        unsigned long long *n_outs, // __device__
        unsigned long long n_chunks,
        unsigned long long *n_reader_chunk, // __managed__, HtoD
        unsigned long long *n_writer_chunk, // __managed__, DtoH
        // input vector
        const R *cfgs, const uint64_t n_cfgs,
        // constants
        uint8_t ea,
        const frow_t *f0L, const uint32_t f0Lsz,
        const frow_t *f0R, const uint32_t f0Rsz) {
    auto idx = threadIdx.x + static_cast<uint64_t>(blockIdx.x) * blockDim.x;
    if (idx >= n_cfgs * f0Lsz * f0Rsz) [[unlikely]] return;
    auto r = cfgs[idx / f0Rsz / f0Lsz];
    auto fL = f0L[idx / f0Rsz % f0Lsz];
    auto fR = f0R[idx % f0Rsz];
    auto cfg = parse_R<H>(r, ea);
    if (fL.shape & ~cfg.empty_area) [[unlikely]] return;
    if (fR.shape & ~cfg.empty_area) [[unlikely]] return;
    if (fL.shape & fR.shape) [[unlikely]] return;
    d_push(cfg.nm_cnt, cfg.ex, fL.nm0123);
    d_push(cfg.nm_cnt, cfg.ex, fR.nm0123);
    d_sn(cfg.nm_cnt, cfg.ex);
    if (!d_uniq_chk(cfg.nm_cnt, cfg.ex)) [[unlikely]] return;
    cfg.empty_area &= ~fL.shape;
    cfg.empty_area &= ~fR.shape;
    auto ocfg = assemble_R<H - 1>(cfg);
    auto out = __nv_atomic_fetch_add(n_outs, 1,
            __NV_ATOMIC_ACQUIRE, __NV_THREAD_SCOPE_DEVICE);
spin:
    auto nrc = __nv_atomic_load_n(n_reader_chunk,
            __NV_ATOMIC_ACQUIRE, __NV_THREAD_SCOPE_SYSTEM);
    if (out >= (nrc + n_chunks - 1u) * CYC_CHUNK) {
        __nanosleep(1000000);
        goto spin;
    }
    ring_buffer[out % (n_chunks * CYC_CHUNK)] = ocfg; // slice
    if (out && out % CYC_CHUNK == 0) {
        auto tgt = out / CYC_CHUNK;
        auto src = tgt - 1;
        while (!__nv_atomic_compare_exchange_n(
                    n_writer_chunk,
                    &src, tgt, /* ignored */ true,
                    __NV_ATOMIC_RELEASE, __NV_ATOMIC_RELAXED,
                    __NV_THREAD_SCOPE_SYSTEM)) {
            if (src >= tgt) __builtin_unreachable();
            src = tgt - 1;
            __nanosleep(1000000);
        }
    }
}

template <typename ... TArgs>
void launch(unsigned b, unsigned t, hipStream_t s, unsigned height,
        TArgs && ... args) {
    if (height == 8)
        d_row_search<8><<<b, t, 0, s>>>(std::forward<TArgs>(args)...);
    else if (height == 7)
        d_row_search<7><<<b, t, 0, s>>>(std::forward<TArgs>(args)...);
    else if (height == 6)
        d_row_search<6><<<b, t, 0, s>>>(std::forward<TArgs>(args)...);
    else if (height == 5)
        d_row_search<5><<<b, t, 0, s>>>(std::forward<TArgs>(args)...);
    else if (height == 4)
        d_row_search<4><<<b, t, 0, s>>>(std::forward<TArgs>(args)...);
    else if (height == 3)
        d_row_search<3><<<b, t, 0, s>>>(std::forward<TArgs>(args)...);
    else if (height == 2)
        d_row_search<2><<<b, t, 0, s>>>(std::forward<TArgs>(args)...);
    else if (height == 1)
        d_row_search<1><<<b, t, 0, s>>>(std::forward<TArgs>(args)...);
    else
        throw std::runtime_error{ std::format("height {} not supported", height) };
}

CudaSearcher::CudaSearcher(uint64_t empty_area)
    : solutions{}, height{ (std::bit_width(empty_area) + 8u - 1u) / 8u } {
    auto &r = solutions[empty_area & 0xffu];
    C(hipMallocManaged(&r.ptr, sizeof(R)));
    r.ptr[0] = RX{ (uint32_t)(empty_area >> 8), (uint32_t)(empty_area >> 8 + 32) };
    r.len = 1;
}

CudaSearcher::~CudaSearcher() {
    free();
}

void CudaSearcher::free() {
    for (auto &r : solutions) {
        if (r.ptr)
            hipFree(r.ptr);
        r.ptr = nullptr;
        r.len = 0;
    }
}

std::pair<uint64_t, uint32_t> balance(uint64_t n) {
    if (n <= 32)
        return { 1, n };
    if (n <= 32 * 84 * 3)
        return { (n + 31) / 32, 32 };
    if (n <= 64 * 84 * 3)
        return { (n + 63) / 64, 64 };
    if (n <= 96 * 84 * 3)
        return { (n + 95) / 96, 96 };
    if (n <= 128 * 84 * 3)
        return { (n + 127) / 128, 128 };
    if (n <= 256 * 84 * 3)
        return { (n + 255) / 256, 256 };
    return { (n + 511) / 512, 512 };
}

struct Device {
    int dev;
    hipStream_t c_stream, m_stream;

    RX *ring_buffer; // __device__
    unsigned long long n_chunks;
    unsigned long long *counters; // __managed__, n_reader_chunk, n_writer_chunk

    unsigned long long *n_outs; // __device__, owned

    uint64_t workload;

    unsigned long long m_scheduled;
    std::deque<Rg<RX>> m_data;
    std::deque<hipEvent_t> m_events;

    explicit Device(int d)
        : dev{ d }, c_stream{}, m_stream{}, ring_buffer{},
          n_chunks{}, counters{}, n_outs{}, workload{},
          m_scheduled{}, m_data{}, m_events{} {

        C(hipMallocManaged(&counters, 2 * sizeof(unsigned long long)));
        cuda::atomic_ref n_reader_chunk{ counters[0] };
        cuda::atomic_ref n_writer_chunk{ counters[1] };
        n_reader_chunk.store(0, cuda::memory_order_release);
        n_writer_chunk.store(0, cuda::memory_order_release);

        C(hipSetDevice(d));

        size_t sz_free, sz_total;
        C(hipMemGetInfo(&sz_free, &sz_total));
        n_chunks = (9 * sz_free / 10 / sizeof(RX) + CYC_CHUNK - 1) / CYC_CHUNK;

        C(hipStreamCreateWithFlags(&c_stream, hipStreamNonBlocking));
        C(hipStreamCreateWithFlags(&m_stream, hipStreamNonBlocking));

        C(hipMallocAsync(&n_outs, sizeof(unsigned long long), c_stream));
        unsigned long long zero{};
        C(hipMemcpyAsync(n_outs, &zero, sizeof(zero), hipMemcpyHostToDevice, c_stream));

        std::cout << std::format("dev#{}: allocating {} * {}B = {}B ring buffer\n",
                dev, n_chunks, display(CYC_CHUNK * sizeof(RX)),
                display(n_chunks * CYC_CHUNK * sizeof(RX)));
        C(hipMallocAsync(&ring_buffer, n_chunks * CYC_CHUNK * sizeof(RX), c_stream));
    }

    ~Device() {
        C(hipSetDevice(dev));
        for (auto ev : m_events)
            C(hipEventDestroy(ev));
        C(hipStreamSynchronize(c_stream));
        C(hipStreamDestroy(c_stream));
        C(hipStreamSynchronize(m_stream));
        C(hipStreamDestroy(m_stream));
        C(hipFree(n_outs));
        C(hipFree(counters));
        C(hipFree(ring_buffer));
    }

    [[nodiscard]] bool c_completed() const {
        auto res = hipStreamQuery(c_stream);
        switch (res) {
            case hipSuccess: return true;
            case hipErrorNotReady: return false;
            default: C(res); return false;
        }
    }

    [[nodiscard]] bool m_completed() const {
        return m_data.empty();
    }

    void dispatch(unsigned pos, unsigned height, Rg<R> cfgs) {
        auto [ptr, len] = cfgs;
        if (!ptr || !len)
            return;
        auto szid = min(height - 1, 5);
        auto fanoutL = h_frowInfoL[(pos >> 0) & 0b1111u].sz[szid];
        auto fanoutR = h_frowInfoR[(pos >> 4) & 0b1111u].sz[szid];
        auto sz = len * fanoutL * fanoutR;
        auto d_f0L = d_frowDataL[dev][pos >> 0 & 0xfu];
        auto d_f0R = d_frowDataR[dev][pos >> 4 & 0xfu];
        auto [b, t] = balance(sz);
        std::cout << std::format("dev#{}: 0b{:08b}<<<{:8}, {:3}>>> = {:<6}*L{:<5}*R{:<5} => {:>9}B\n",
                dev, pos, b, t,
                len, fanoutL, fanoutR, display(sz * sizeof(R)));
        C(hipSetDevice(dev));
        C(hipMemAdvise(ptr, len * sizeof(R), hipMemAdviseSetReadMostly, dev));
        C(hipStreamAttachMemAsync(c_stream, ptr, len * sizeof(R)));
        C(hipMemPrefetchAsync(ptr, len * sizeof(R), dev, c_stream));
        launch(b, t, c_stream, height,
                // output ring buffer
                ring_buffer, n_outs,
                n_chunks,
                &counters[0], &counters[1],
                // input vector
                ptr, len,
                // constants
                pos,
                d_f0L, fanoutL,
                d_f0R, fanoutR);
        workload += b * t;
    }

    void recycle(bool last) {
        C(hipSetDevice(dev));

        if (last) {
            std::cout << std::format("dev#{}: synchronize\n", dev);
            C(hipStreamSynchronize(c_stream)); // necessary as kernels may be still finishing
        }

        cuda::atomic_ref n_reader_chunk{ counters[0] };
        cuda::atomic_ref n_writer_chunk{ counters[1] };
        auto nwc = n_writer_chunk.load(cuda::memory_order_acquire);
        while (m_scheduled < nwc) {
            std::cout << std::format("dev#{}: start DtoH chunk #{:0{}}/{} ({} B)\n",
                    dev, m_scheduled, count_digits(n_chunks),
                    n_chunks, display(CYC_CHUNK * sizeof(RX)));
            Rg<RX> r{ new RX[CYC_CHUNK], CYC_CHUNK };
            C(hipMemcpyAsync(r.ptr,
                        ring_buffer + (m_scheduled % n_chunks) * CYC_CHUNK,
                        CYC_CHUNK * sizeof(RX), hipMemcpyDeviceToHost, m_stream));
            hipEvent_t ev;
            C(hipEventCreateWithFlags(&ev, hipEventDisableTiming));
            C(hipEventRecord(ev, m_stream));
            m_data.push_back(r);
            m_events.push_back(ev);
            m_scheduled++;
        }

        if (!last)
            return;

        unsigned long long used;
        C(hipMemcpyAsync(&used, n_outs, sizeof(used), hipMemcpyDeviceToHost, c_stream));
        C(hipStreamSynchronize(c_stream));
        if (used < nwc * CYC_CHUNK)
            throw std::runtime_error{ "internal error" };
        if (used >= (nwc + 1u) * CYC_CHUNK)
            throw std::runtime_error{ "internal error" };
        if (m_scheduled != nwc)
            throw std::runtime_error{ "internal error" };
        if (used == nwc * CYC_CHUNK)
            return;

        auto sz = used - nwc * CYC_CHUNK;
        std::cout << std::format("dev#{}: start tail DtoH chunk #{:0{}}/{} for {} entries ({} B)\n",
                dev, nwc, count_digits(n_chunks),
                n_chunks, sz, display(sz * sizeof(RX)));
        Rg<RX> r{ new RX[sz], sz };
        C(hipMemcpyAsync(r.ptr, ring_buffer + (nwc % n_chunks) * CYC_CHUNK,
                    sz * sizeof(RX), hipMemcpyDeviceToHost, m_stream));
        hipEvent_t ev;
        C(hipEventCreateWithFlags(&ev, hipEventDisableTiming));
        C(hipEventRecord(ev, m_stream));
        m_data.push_back(r);
        m_events.push_back(ev);
    }

    void collect(Sorter &sorter) {
        cuda::atomic_ref n_reader_chunk{ counters[0] };
        while (!m_events.empty()) {
            auto ev = m_events.front();
            auto err = hipEventQuery(ev);
            if (err == hipErrorNotReady)
                return;
            C(err);
            C(hipEventDestroy(ev));
            auto nrc = n_reader_chunk.fetch_add(1, cuda::memory_order_release);
            std::cout << std::format("dev#{}: pushing chunk #{:0{}} ({} entries, {} B) to sorter\n",
                    dev, nrc, count_digits(n_chunks),
                    CYC_CHUNK, display(CYC_CHUNK * sizeof(RX)));
            sorter.push(m_data.front());
            m_events.pop_front();
            m_data.pop_front();
        }
    }
};

void CudaSearcher::search_GPU() {
    Sorter sorter{ *this };
    std::vector<std::unique_ptr<Device>> devs;
    for (auto i = 0; i < n_devices; i++)
        devs.emplace_back(std::make_unique<Device>(i));

    for (auto ipos = 0u; ipos <= 255u; ipos++) {
        std::ranges::sort(devs, std::greater{}, [](const std::unique_ptr<Device> &dev) {
            return dev->workload;
        });
        devs.front()->dispatch(ipos, height, solutions[ipos]);
        for (auto &dev : devs) {
            dev->recycle(false);
            dev->collect(sorter);
        }
    }
    bool flag;
    do {
        flag = true;
        for (auto &dev : devs) {
            flag &= dev->c_completed();
            dev->recycle(false);
            dev->collect(sorter);
        }
    } while (!flag);
    for (auto &dev : devs) {
        dev->recycle(true);
        dev->collect(sorter);
    }
    do {
        flag = true;
        for (auto &dev : devs) {
            flag &= dev->m_completed();
            dev->collect(sorter);
        }
    } while (!flag);
    devs.clear();
    sorter.join();
    height--;
}

uint64_t CudaSearcher::next_size(unsigned pos) const {
    auto szid = min(height - 1, 5);
    return solutions[pos].len
        * h_frowInfoL[(pos >> 0) & 0b1111u].sz[szid]
        * h_frowInfoR[(pos >> 4) & 0b1111u].sz[szid];
}

Rg<R> CudaSearcher::write_solution(unsigned pos, size_t sz) {
    auto &r = solutions[pos];
    if (r.ptr) {
        C(hipFree(r.ptr));
        r.ptr = nullptr, r.len = 0;
    }
    if (sz)
        C(hipMallocManaged(&r.ptr, sz * sizeof(R), hipMemAttachHost));
    r.len = sz;
    return r;
}

Rg<R> *CudaSearcher::write_solutions(size_t sz) {
    for (auto pos = 0; pos <= 255; pos++) {
        auto &[ptr, len] = solutions[pos];
        if (ptr) C(hipFree(ptr));
        ptr = nullptr;
        len = 0;
        C(hipMallocManaged(&ptr, sz * sizeof(R), hipMemAttachHost));
    }
    return solutions;
}
