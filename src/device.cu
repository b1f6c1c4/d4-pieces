#include "hip/hip_runtime.h"
#include "device.h"

#include <cuda/atomic>
#include <chrono>
#include <sys/sysinfo.h>
#include <pthread.h>
#include <unistd.h>
#include "util.hpp"
#include "util.cuh"
#include "kernel.h"

using namespace std::chrono_literals;

Device::Input::Input(WL work, int dev, unsigned height)
    : WL{ work }, szid{ min(height - 1, 5) }, kp{ KSizing{ len,
        h_frowInfoL[pos >> 0 & 0b1111u].sz[szid],
        h_frowInfoR[pos >> 4 & 0b1111u].sz[szid] }.optimize(true) },
      p{} { }

Device::Device(int d, unsigned h, Sorter &s)
    : dev{ d }, height{ h }, sorter{ s } {
    C(hipSetDevice(d));

    C(hipMallocManaged(&counters, 2 * sizeof(unsigned long long)));
    cuda::atomic_ref n_reader_chunk{ counters[0] };
    cuda::atomic_ref n_writer_chunk{ counters[1] };
    n_reader_chunk.store(0, cuda::memory_order_release);
    n_writer_chunk.store(0, cuda::memory_order_release);

    size_t sz_free, sz_total;
    C(hipMemGetInfo(&sz_free, &sz_total));
    n_chunks = (7 * sz_free / 10 / sizeof(RX) + CYC_CHUNK - 1) / CYC_CHUNK;

    // launch thread AFTER setting up counters and n_chunks
    c_thread = std::thread{ &Device::c_entry, this };
    m_thread = std::thread{ &Device::m_entry, this };
}

void Device::c_entry() {
    pthread_setname_np(pthread_self(), std::format("dev#{}.c", dev).c_str());
    C(hipSetDevice(dev));

    C(hipStreamCreateWithFlags(&c_stream, hipStreamNonBlocking));

    C(hipMallocAsync(&n_outs, sizeof(unsigned long long), c_stream));
    C(hipMemsetAsync(n_outs, 0, 2 * sizeof(unsigned long long), c_stream));

    std::cout << std::format("dev#{}.c: allocating {} * {}B = {}B ring buffer\n",
            dev, n_chunks, display(CYC_CHUNK * sizeof(RX)),
            display(n_chunks * CYC_CHUNK * sizeof(RX)));
    C(hipMallocAsync(&ring_buffer, n_chunks * CYC_CHUNK * sizeof(RX), c_stream));

    std::unique_lock lock{ mtx };
    boost::upgrade_lock lock_c_works{ mtx_c };

    xc_ready = true;
    cv.notify_all();

again:
    cv.wait_for(lock, 50ms, [this]{ return xc_closed || !xc_queue.empty(); });

    // synchronously free up original copies (work.ptr)
    lock.unlock();

    for (auto &work : c_works) {
        if (work.ev_m == hipEvent_t{})
            continue;
        auto err = hipEventQuery(work.ev_m);
        if (err == hipErrorNotReady)
            continue;
        C(err);
        {
            // since indefinite long time might have passed before actual
            // kernel start and ev_m check, this is only a rough estimate
            boost::upgrade_to_unique_lock xlock_c_work{ lock_c_works };
            work.est_start_time = std::chrono::steady_clock::now();
        }
        C(hipEventDestroy(work.ev_m));
        work.ev_m = hipEvent_t{};
        if (!work.device_accessible()) {
            std::cout << std::format("dev#{}.c: free up {}B host input mem ({} entries)\n",
                    dev, display(work.len * sizeof(R)), work.len);
            work.dispose();
        }
    }

    // synchronously free up device copies (work.p)
    while (!c_works.empty()) {
        auto work = c_works.front(); // copy; pop_front() anyway
        auto err = hipEventQuery(work.ev_c);
        if (err == hipErrorNotReady)
            break;
        C(err);
        C(hipEventDestroy(work.ev_c));
        work.ev_c = hipEvent_t{};
        std::cout << std::format("dev#{}.c: free up {}B device mem ({} entries)\n",
                dev, display(work.len * sizeof(R)), work.len);
        if (work.device_accessible())
            work.dispose();
        else {
            C(hipFree(work.p));
            work.p = nullptr;
        }
        {
            boost::upgrade_to_unique_lock xlock_c_work{ lock_c_works };
            c_works.pop_front();
            c_fom_done += work.kp.fom();
            c_actual_done += work.elapsed();
        }
        std::cout << std::format("dev#{}.c: {}B ({} entries) device mem freed\n",
                dev, display(work.len * sizeof(R)), work.len);
    }

    // launch kernels
    lock.lock();
    if (!xc_queue.empty()) { // make sure not to issue too many
        auto work = std::move(xc_queue.front());
        xc_queue.pop_front();
        lock.unlock();
        { // dispatch logic
            KParamsFull kpf{ work.kp, height,
                ring_buffer, n_outs, n_chunks,
                &counters[0], &counters[1],
                work.ptr, (uint8_t)work.pos,
                d_frowDataL[dev][work.pos >> 0 & 0xfu],
                d_frowDataR[dev][work.pos >> 4 & 0xfu] };
            std::cout << std::format(
                    "dev#{}.c: {:08b}<<<{:8},{:4},{:5}B>>>[{}] = {:<6}*L{:<5}*R{:<5} ~ {}\n",
                    dev, work.pos,
                    kpf.blocks, kpf.threads, kpf.shmem_len * sizeof(frow32_t),
                    !kpf.shmem_len ? "-" : kpf.reverse ? "L" : "R",
                    kpf.n_cfgs, kpf.f0Lsz, kpf.f0Rsz, display(kpf.fom()));
            if (!work.device_accessible()) {
                C(hipMallocAsync(&work.p, work.len * sizeof(R), c_stream));
                C(hipMemcpyAsync(work.p, work.ptr, work.len * sizeof(R),
                            hipMemcpyHostToDevice, c_stream));
                kpf.cfgs = work.p;
            } else {
                C(hipMemAdvise(work.ptr, work.len * sizeof(R), hipMemAdviseSetReadMostly, dev));
                C(hipMemPrefetchAsync(work.ptr, work.len * sizeof(R), dev, c_stream));
            }
            C(hipEventCreateWithFlags(&work.ev_m, hipEventDisableTiming));
            C(hipEventRecord(work.ev_m, c_stream));
            kpf.launch(c_stream);
            C(hipEventCreateWithFlags(&work.ev_c, hipEventDisableTiming));
            C(hipEventRecord(work.ev_c, c_stream));

            boost::upgrade_to_unique_lock xlock_c_work{ lock_c_works };
            c_sum_fom += work.kp.fom();
            c_works.emplace_back(work);
        }
        lock.lock();
    }
    if (!xc_closed || !c_works.empty())
        goto again;

    lock.unlock();
    unsigned long long used;
    { // figure out final output count
        C(hipMemcpyAsync(&used, n_outs, sizeof(used), hipMemcpyDeviceToHost, c_stream));
        C(hipStreamSynchronize(c_stream));
        C(hipStreamDestroy(c_stream));
        c_stream = hipStream_t{};
        C(hipFree(n_outs));
        n_outs = nullptr;
    }
    lock.lock();
    std::cout << std::format("dev#{}.c: c thread quitting xc_used={}\n", dev, used);
    xc_used = used;
    cv.notify_all();
}

#define MAX_DtoH 8

void Device::m_entry() {
    pthread_setname_np(pthread_self(), std::format("dev#{}.m", dev).c_str());
    C(hipSetDevice(dev));

    C(hipStreamCreateWithPriority(&m_stream, hipStreamNonBlocking, -1));

    cuda::atomic_ref n_reader_chunk{ counters[0] };
    cuda::atomic_ref n_writer_chunk{ counters[1] };

    std::unique_lock lock{ mtx };
    boost::upgrade_lock lock_m_works{ mtx_m };

    cv.wait(lock, [this]{ return xc_ready; });

    auto tailed = false;

again:
    cv.wait_for(lock, 50ms, [this]{ return xc_used.has_value(); });
    auto used = std::move(xc_used);
    if (xc_used) {
        tailed = true;
        xc_used.reset();
    }
    lock.unlock();

again2:
    // sorter logic
    auto local = 0ull;
    while (!m_works.empty()) {
        auto pwork = m_works.front();
        std::atomic_ref ptr{ pwork->ptr };
        if (!ptr.load(std::memory_order_relaxed)) {
            boost::upgrade_to_unique_lock xlock_m_works{ lock_m_works };
            m_works.pop_front();
            local++;
        } else {
            break;
        }
    }
    if (local) {
        n_reader_chunk.fetch_add(local, cuda::memory_order_release);
    }

    // recycle logic
    auto nwc = n_writer_chunk.load(cuda::memory_order_acquire);
    while (m_scheduled < nwc && m_works.size() < MAX_DtoH) {
        m_initiate_transfer(CYC_CHUNK, lock_m_works);
    }
    if (m_works.size() >= MAX_DtoH) {
        std::this_thread::sleep_for(5ms);
        goto again2;
    }

    if (used) { // tail recycle logic
        if (*used < nwc * CYC_CHUNK)
            throw std::runtime_error{ "internal error" };
        if (*used >= (nwc + 1u) * CYC_CHUNK)
            throw std::runtime_error{ "internal error" };
        if (m_scheduled != nwc)
            throw std::runtime_error{ "internal error" };
        if (*used > nwc * CYC_CHUNK) {
            m_initiate_transfer(*used - nwc * CYC_CHUNK, lock_m_works);
        }
    }

    lock.lock();
    if (!m_works.empty() || !tailed)
        goto again;

    xm_completed = true;
    cv.notify_all();
    lock.unlock();
    std::cout << std::format("dev#{}.m: m thread quitting\n", dev);
    C(hipFree(ring_buffer));
    ring_buffer = nullptr;
    C(hipFree(counters));
    counters = nullptr;
}

void Device_callback_helper(void *raw) {
    auto data = reinterpret_cast<void **>(raw);
    auto *self = static_cast<Device *>(data[0]);
    auto *pwork = static_cast<Device::Output *>(data[1]);
    self->m_callback(pwork);
    delete [] data;
}

void Device::m_initiate_transfer(uint64_t sz, boost::upgrade_lock<boost::upgrade_mutex> &lock) {
    std::cout << std::format("dev#{}.m: start {}DtoH chunk #{:0{}}/{} {} ({}B)\n",
            dev, sz == CYC_CHUNK ? "" : "tail ",
            m_scheduled, count_digits(n_chunks),
            n_chunks, sz, display(sz * sizeof(RX)));
    Output *pwork;
    {
        boost::upgrade_to_unique_lock xlock_m_works{ lock };
        pwork = m_works.emplace_back(new Output{ Rg<RX>::make_cpu(sz) });
        m_scheduled++;
    }
    C(hipMemcpyAsync(pwork->ptr,
                ring_buffer + ((m_scheduled - 1) % n_chunks) * CYC_CHUNK,
                sz * sizeof(RX), hipMemcpyDeviceToHost, m_stream));
    auto data = new void *[2]{ this, pwork };
    C(hipLaunchHostFunc(m_stream, &Device_callback_helper, data));
}

void Device::m_callback(Output *pwork) {
    std::cout << std::format("dev#{}.m: pushing a chunk ({} entries, {}B) to sorter\n",
            dev, pwork->len, display(pwork->len * sizeof(RX)));
    sorter.push(*pwork);
    std::atomic_ref ptr{ pwork->ptr };
    ptr.store(nullptr, std::memory_order_relaxed);
}

void Device::dispatch(WL cfgs) {
    if (!cfgs.ptr || !cfgs.len)
        return;
    std::unique_lock lock{ mtx };
    cv.wait(lock, [this]{ return xc_ready; });
    auto &work = xc_queue.emplace_back(cfgs, dev, height);
    c_fom_queued.fetch_add(work.kp.fom(), std::memory_order_relaxed);
    cv.notify_all();
}

void Device::close() {
    std::unique_lock lock{ mtx };
    xc_closed = true;
    cv.notify_all();
}

void Device::wait() {
    std::unique_lock lock{ mtx };
    cv.wait(lock, [this]{ return xm_completed; });
}

double Device::get_etc() const {
    boost::shared_lock lock_c_works{ mtx_c };
    if (c_works.empty())
        return 0;

    auto &work = c_works.front();
    auto el = work.elapsed();
    auto q = c_fom_queued.load(std::memory_order_relaxed);
    return q + c_sum_fom - std::min(work.kp.fom(), el);
}

unsigned Device::print_stats() const {
    cuda::atomic_ref n_reader_chunk{ counters[0] };
    cuda::atomic_ref n_writer_chunk{ counters[1] };

    auto lines = 1u;
    std::stringstream ss;
    ss << "\33[37mdev" << dev << " [";

    std::unique_lock lock{ mtx };
    if (!xc_ready) {
        ss << "initializing]";
    } else if (xm_completed) {
        ss << "completed]";
    } else {
        lock.unlock();
        auto nwc = n_writer_chunk.load(cuda::memory_order_relaxed);

        boost::shared_lock lock_m_works{ mtx_m };
        auto nrc = n_reader_chunk.load(cuda::memory_order_relaxed);
        for (auto i = 0ull; i < n_chunks; i++) {
            auto c = i < nrc ? i + n_chunks : i;
            if (c < nrc)
                ss << " ";
            else if (c < m_scheduled) {
                if (m_works[c - nrc]->ptr)
                    ss << "\33[35mR";
                else
                    ss << "\33[95mR";
            } else if (c < nwc)
                ss << "\33[90m-";
            else if (c == nwc)
                ss << "\33[36mW";
            else
                ss << " ";
        }
        lock_m_works.unlock();

        ss << "\33[37m] ";

        boost::shared_lock lock_c_works{ mtx_c };
        ss << std::format("[E{:7} A{:7}]",
                display(c_fom_done), display(c_actual_done));
        ss << std::format(" ETC{:7}", display(get_etc()));

        lock.lock();
        if (!xc_queue.empty())
            ss << std::format(" Q{:d}", xc_queue.size());
        if (xc_closed)
            ss << " closed";

        if (!c_works.empty()) {
            auto &work = c_works.front();
            lines++;
            ss << std::format("\33[K\n\33[37mdev#{} {:08b}{}/{:7}]",
                        dev, work.pos, work.kp.to_string(true), display(work.elapsed()));
            if (c_works.size() >= 1)
                ss << " + W" << c_works.size() - 1;
        }
    }
    lock.unlock();
    ss << "\33[K\33[0m\n";
    std::cerr << ss.str();
    return lines;
}

Device::~Device() {
    // it is necessary to explicitly join c/m_thread now, because implicitly
    // defined destructors will destroy data members in the reverse order of
    // declaration and clash the execution of tail
    c_thread.join();
    m_thread.join();
}
