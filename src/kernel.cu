#include "hip/hip_runtime.h"
#include "kernel.h"

#include <algorithm>
#include <ranges>
#include <vector>
#include <format>
#include <iostream>

#include "util.cuh"
#include "util.hpp"

template <unsigned H, int>
__global__
void legacy_row_search(unsigned, K_PARAMS);

template <unsigned H, int W, bool Reverse>
__global__
__launch_bounds__(W, 1536 / W)
void LR_row_search(unsigned shmem_len, K_PARAMS);

extern template __global__ void legacy_row_search<8, 0>(unsigned, K_PARAMS);
extern template __global__ void legacy_row_search<7, 0>(unsigned, K_PARAMS);
extern template __global__ void legacy_row_search<6, 0>(unsigned, K_PARAMS);
extern template __global__ void legacy_row_search<5, 0>(unsigned, K_PARAMS);
extern template __global__ void legacy_row_search<4, 0>(unsigned, K_PARAMS);
extern template __global__ void legacy_row_search<3, 0>(unsigned, K_PARAMS);
extern template __global__ void legacy_row_search<2, 0>(unsigned, K_PARAMS);
extern template __global__ void legacy_row_search<1, 0>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<8, 768, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<7, 768, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<6, 768, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<5, 768, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<4, 768, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<3, 768, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<2, 768, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<1, 768, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<8, 768, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<7, 768, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<6, 768, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<5, 768, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<4, 768, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<3, 768, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<2, 768, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<1, 768, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<8, 1024, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<7, 1024, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<6, 1024, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<5, 1024, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<4, 1024, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<3, 1024, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<2, 1024, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<1, 1024, true>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<8, 1024, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<7, 1024, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<6, 1024, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<5, 1024, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<4, 1024, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<3, 1024, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<2, 1024, false>(unsigned, K_PARAMS);
extern template __global__ void LR_row_search<1, 1024, false>(unsigned, K_PARAMS);

#define COMMA ,
#define CCMMA ,

void KParamsFull::launch(hipStream_t stream) {
#define ARGS \
    shmem_len, \
    ring_buffer, n_outs, n_chunks, \
    n_reader_chunk, n_writer_chunk, \
    cfgs, n_cfgs, \
    ea, f0L, f0Lsz, f0R, f0Rsz

#define L(k, t) \
    do { if (height == 8) k<8, t><<<blocks, threads, shmem_len * sizeof(frow32_t), stream>>>(ARGS); \
    else if (height == 7) k<7, t><<<blocks, threads, shmem_len * sizeof(frow32_t), stream>>>(ARGS); \
    else if (height == 6) k<6, t><<<blocks, threads, shmem_len * sizeof(frow32_t), stream>>>(ARGS); \
    else if (height == 5) k<5, t><<<blocks, threads, shmem_len * sizeof(frow32_t), stream>>>(ARGS); \
    else if (height == 4) k<4, t><<<blocks, threads, shmem_len * sizeof(frow32_t), stream>>>(ARGS); \
    else if (height == 3) k<3, t><<<blocks, threads, shmem_len * sizeof(frow32_t), stream>>>(ARGS); \
    else if (height == 2) k<2, t><<<blocks, threads, shmem_len * sizeof(frow32_t), stream>>>(ARGS); \
    else if (height == 1) k<1, t><<<blocks, threads, shmem_len * sizeof(frow32_t), stream>>>(ARGS); \
    else throw std::runtime_error{ std::format("height {} not supported", height) }; \
    } while (false)

    if (!shmem_len) L(legacy_row_search, 0);
    else if (threads > 768)
        if (reverse) L(LR_row_search, 1024 COMMA true);
        else L(LR_row_search, 1024 COMMA false);
    else
        if (reverse) L(LR_row_search, 768 COMMA true);
        else L(LR_row_search, 768 COMMA false);
}

void prepare_kernels() {
#define S(...) \
    C(hipFuncSetAttribute(reinterpret_cast<const void*>(__VA_ARGS__), hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared)); \
    C(hipFuncSetAttribute(reinterpret_cast<const void*>(__VA_ARGS__), hipFuncAttributeMaxDynamicSharedMemorySize, 101376));
#define SS(k, ...) \
    S(k<8 COMMA __VA_ARGS__>) \
    S(k<7 COMMA __VA_ARGS__>) \
    S(k<6 COMMA __VA_ARGS__>) \
    S(k<5 COMMA __VA_ARGS__>) \
    S(k<4 COMMA __VA_ARGS__>) \
    S(k<3 COMMA __VA_ARGS__>) \
    S(k<2 COMMA __VA_ARGS__>) \
    S(k<1 COMMA __VA_ARGS__>)
    SS(legacy_row_search, 0)
    SS(LR_row_search, 768  COMMA true)
    SS(LR_row_search, 768  COMMA false)
    SS(LR_row_search, 1024 COMMA true)
    SS(LR_row_search, 1024 COMMA false)
}

static unsigned known_t[]{ 96, 128, 192, 256, 384, 512, 768, 1024 };
static unsigned known_shmem_b[]{ 5120, 7168, 11776, 15872, 24576, 32768, 50176, 101376 };

#ifdef BMARK
std::vector<KParams> KSizing::optimize() const {
#else
KParams KSizing::optimize(bool debug) const {
#endif
    std::vector<KParams> pars;
    auto n = n_cfgs * f0Lsz * f0Rsz;
    for (auto t = 1ull; t <= 512u; t <<= 1)
        if ((n + t - 1) / t <= 2147483647ull)
            pars.emplace_back(*this, false, (n + t - 1) / t, t, 0);
    for (auto t = 3ull; t <= 1024u; t <<= 1)
        if ((n + t - 1) / t <= 2147483647ull)
            pars.emplace_back(*this, false, (n + t - 1) / t, t, 0);
    auto wpn = (n_cfgs + 31) / 32;
    for (auto i = 0; i < 8; i++) {
        for (auto b = 1ull; b <= wpn && b <= 2147483647ull; b <<= 1) {
            pars.emplace_back(*this, false, b, known_t[i], known_shmem_b[i] / sizeof(frow32_t));
            pars.emplace_back(*this, true, b, known_t[i], known_shmem_b[i] / sizeof(frow32_t));
        }
        for (auto b = 3ull; b <= wpn && b <= 2147483647ull; b <<= 1) {
            pars.emplace_back(*this, false, b, known_t[i], known_shmem_b[i] / sizeof(frow32_t));
            pars.emplace_back(*this, true, b, known_t[i], known_shmem_b[i] / sizeof(frow32_t));
        }
        for (auto b = 7ull; b <= wpn && b <= 2147483647ull; b <<= 1) {
            pars.emplace_back(*this, false, b, known_t[i], known_shmem_b[i] / sizeof(frow32_t));
            pars.emplace_back(*this, true, b, known_t[i], known_shmem_b[i] / sizeof(frow32_t));
        }
        for (auto b = 21ull; b <= wpn && b <= 2147483647ull; b <<= 1) {
            pars.emplace_back(*this, false, b, known_t[i], known_shmem_b[i] / sizeof(frow32_t));
            pars.emplace_back(*this, true, b, known_t[i], known_shmem_b[i] / sizeof(frow32_t));
        }
    }
    std::ranges::sort(pars, std::less{}, [](const KParams &kp) { return kp.fom(); });
#ifdef BMARK
    return pars;
#else
    if (debug) {
        std::cout << std::format("kernel#optimize: best kernel params for {} are:\n",
                to_string());
        for (auto i = 0zu; i < pars.size() && i < 10zu; i++)
            std::cout << std::format("      #{}\n", pars[i].to_string(false));
    }
    return pars.front();
#endif
}

std::string KSizing::to_string() const {
    return std::format("[{:<6}*L{:<5}*R{:<5}]", n_cfgs, f0Lsz, f0Rsz);
}

std::string KParams::to_string(bool full) const {
    std::string s;
    if (!shmem_len)
        s = std::format("<<<{:11},{:5}>>>[legacy]", blocks, threads);
    else
        s = std::format("<<<{:9},{:5},{:6}>>>[{}]", blocks, threads,
                shmem_len * sizeof(frow32_t), reverse ? 'L' : 'R');
    if (full)
        s += KSizing::to_string();
    s += std::format(" ~ {}", display(fom()));
    return s;
}

#ifdef BMARK
double KParams::fom(bool debug) const {
#else
double KParams::fom() const {
#endif
    auto oc = std::min(16u, 1536u / threads) * 84; // max blocks per device
    auto util = 1536.0 / ((1536u / threads) * threads);
    auto e = ((blocks + oc - 1) / oc);

    if (shmem_len == 0) {
        auto c = (1.0 + ((threads + 31) / 32 * 32) * 1e-3) * 1.63e-6;
        auto v = e * c + blocks * 1e-11;
#ifdef BMARK
        if (debug) {
            std::cout << std::format("<<<{:10},{:5}>>>  [legacy] {:9.2e}*{:3} + {:9.2f} ={:9.2f}\n",
                    blocks, threads,
                    c, e, blocks * 1e-11, v);
        }
#endif
        return v + 500e-6;
    }

    uint32_t Ltile, Rtile;
    if (f0Lsz + f0Rsz <= shmem_len) {
        Ltile = f0Lsz, Rtile = f0Rsz;
    } else if (f0Lsz < shmem_len / 2) {
        Ltile = f0Lsz, Rtile = shmem_len - f0Lsz;
    } else if (f0Rsz < shmem_len / 2) {
        Ltile = shmem_len - f0Rsz, Rtile = f0Rsz;
    } else {
        Ltile = shmem_len / 2, Rtile = shmem_len - Ltile;
    }
    auto nL = (f0Lsz + Ltile - 1) / Ltile;
    auto nR = (f0Rsz + Rtile - 1) / Rtile;
    if (reverse) {
        std::swap(Ltile, Rtile);
        std::swap(nL, nR);
    }

    auto tpb = static_cast<uint64_t>(threads);
    auto tpg = static_cast<uint64_t>(blocks) * tpb;
    auto iterations = (n_cfgs + tpg - 1) / tpg;

    auto mem = 1.2e-4;
    auto m = nL * (4e-3 + Ltile * mem); // load Lcache
    if (nR == 1) // load Rcache
        m += (4e-3 + Rtile * mem);
    else
        m += nL * nR * (4e-3 + Rtile * mem);

    auto c = nL * nR * Ltile * Rtile * iterations * 7.2e-8; // compute
    auto n = n_cfgs * 2.3e-5; // load cfgs
    auto v = e * (m + c * util) + n;
#ifdef BMARK
    if (debug) {
        std::cout << std::format("<<<{:9},{:5},{:6}>>>   {}{}*{}-{}{}*{}   ({:9.2f} +{:9.2f}*{})*{:3}+{:9.2f}={:9.2f}\n",
                blocks, threads, shmem_len * sizeof(frow32_t),
                reverse ? "R" : "L",
                Ltile,
                nL,
                reverse ? "L" : "R",
                Rtile,
                nR,
                m, c, util, e, n, v);
    }
#endif
    return v + 500e-6;
}
