#include "hip/hip_runtime.h"
#include "growable.cuh"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <algorithm>
#include <format>
#include <iostream>

#define C(ans) { chk_impl((ans), __FILE__, __LINE__); }

static inline void chk_impl(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        throw std::runtime_error{
            std::format("CUDA: {}: {} @ {}:{}\n",
                    hipGetErrorName(code), hipGetErrorString(code),
                    file, line) };
    }
}

static inline void chk_impl(hipError_t code, const char *file, int line) {
    const char *pn = "???", *ps = "???";
    hipDrvGetErrorName(code, &pn);
    hipDrvGetErrorString(code, &ps);
    if (code != hipSuccess) {
        throw std::runtime_error{
            std::format("CUDA Driver: {}: {} @ {}:{}\n", pn, ps, file, line) };
    }
}

static inline void chk_impl(hiprandStatus_t code, const char *file, int line) {
    if (code != HIPRAND_STATUS_SUCCESS) {
        throw std::runtime_error{
            std::format("hiprand: {} @ {}:{}\n", (int)code, file, line) };
    }
}


template <typename T>
Growable<T>::Growable(size_t max)
    : reserved{}, vmaps{}, offset{}, used{}, maps{},
      mapped{}, evicted_data{}, evicted{}, chunk{},
      prop{}, adesc{} {

    int n; C(hipGetDeviceCount(&n)); // dark magic; don't touch

    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    C(hipGetDevice(&prop.location.id));
    adesc.location = prop.location;
    adesc.flags = hipMemAccessFlagsProtReadWrite;

    C(hipMemGetAllocationGranularity(&chunk, &prop, hipMemAllocationGranularityMinimum));
    chunk = (chunk + sizeof(T) - 1) / sizeof(T);
    remap(max);
};

template <typename T>
Growable<T>::~Growable() {
    for (auto rh : maps) {
        C(hipMemUnmap((hipDeviceptr_t)rh.ptr, rh.len * sizeof(T)));
        C(hipMemRelease(rh.h));
    }
    for (auto v : vmaps)
        C(hipMemAddressFree((hipDeviceptr_t)v.ptr, v.len * sizeof(T)));
}

template <typename T>
Growable<T>::R Growable<T>::cpu_merge_sort() {
    evict_all();
    R dest{};
    int dgpu;
    C(hipGetDevice(&dgpu));
    C(hipMallocManaged(&dest.ptr, evicted * sizeof(T)));
    C(hipMemAdvise(dest.ptr, evicted * sizeof(T),
                hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
    std::vector<std::pair<size_t, size_t>> heap;
    for (auto i = 0u; auto r : evicted_data) {
        heap.emplace_back(i++, 0);
        std::ranges::sort(r.ptr, r.ptr + r.len, comp, proj);
    }
    auto pproj = [&,this](size_t blk, size_t id) {
        return evicted_data[blk].ptr[id];
    };
    while (!heap.empty()) {
        std::ranges::pop_heap(heap, std::less{}, pproj);
        auto &[blk, id] = heap.back();
        auto val = evicted_data[blk].ptr[id];
        if (!dest.len || dest.ptr[dest.len - 1] != val)
            dest[dest.len++] = val;
        if (++id == evicted_data[blk].len)
            heap.pop_back();
        else
            std::ranges::push_heap(heap, std::less{}, pproj);
    }
    C(hipMemAdvise(dest.ptr, dest.len * sizeof(T),
               hipMemAdviseSetAccessedBy, dev));
    C(hipMemAdvise(dest.ptr, dest.len * sizeof(T),
               hipMemAdviseSetReadMostly, dev));
    return dest;
}

template <typename T>
bool Growable<T>::ensure(size_t n) {
    if (mapped && used + n <= mapped) return true;

    auto sz = std::max((used + n - mapped + chunk - 1) / chunk, 1zu) * chunk;
    remap(mapped + sz); // vmap must not be empty

    hipMemGenericAllocationHandle_t h;
    hipError_t err;
    if ((err = hipMemCreate(&h, sz * sizeof(T), &prop, 0)) == hipSuccess) goto map;
    if (err != hipErrorOutOfMemory) C(err);
    if (!mapped) goto fail;
again:
    evict1();
    if (used + n <= mapped) return true;
    sz = std::max((used + n - mapped + chunk - 1) / chunk, 1zu) * chunk;
    if ((err = hipMemCreate(&h, sz * sizeof(T), &prop, 0)) == hipSuccess) goto map;
    if (used && err == hipErrorOutOfMemory) goto again;
    if (err != hipErrorOutOfMemory) C(err);
fail:
    evict_all();
    return false;
map:
    auto ptr = vmaps[0].ptr + offset + mapped;
    C(hipMemMap((hipDeviceptr_t)ptr, sz * sizeof(T), 0, h, 0));
    C(hipMemSetAccess((hipDeviceptr_t)ptr, sz * sizeof(T), &adesc, 1));
    maps.emplace_back(RH{ R{ ptr, sz }, h });
    mapped += sz;
    return true;
}

template <typename T>
void Growable<T>::evict1() {
    if (maps.empty())
        throw std::runtime_error{ "cannot evict: nothing was allocated" };

    auto src = maps.front();
    auto used1 = min(used, src.len);
    if (used1) {
        auto dst = evicted_data.emplace_back(R{ new T[used1], used1 });
        if (!dst.ptr)
            throw std::runtime_error{ std::format("new T[{}] failed ({} MiB)", used1, used1 * sizeof(T) / 1048576.0) };
        C(hipMemcpy(dst.ptr, src.ptr, used1 * sizeof(T), hipMemcpyDeviceToHost));
        evicted += used1;
    }
    if (used < src.len) {
        used = 0;
    } else {
        remap(mapped + src.len);
        src = maps.front();
        C(hipMemUnmap((hipDeviceptr_t)src.ptr, src.len * sizeof(T)));
        offset += src.len;
        C(hipMemMap((hipDeviceptr_t)src.ptr, src.len * sizeof(T), 0, src.h, 0));
        C(hipMemSetAccess((hipDeviceptr_t)src.ptr, src.len * sizeof(T), &adesc, 1));
        used -= src.len;
    }
}

template <typename T>
void Growable<T>::compact() {
    if (maps.empty()) {
        cleanup();
        return;
    }
    auto beg = std::ranges::lower_bound(maps, maps[0].ptr + used, std::less{}, &RH::ptr);
    for (auto it = beg; it != maps.end(); it++) {
        C(hipMemUnmap((hipDeviceptr_t)it->ptr, it->len * sizeof(T)));
        mapped -= it->len;
        C(hipMemRelease(it->h));
    }
    maps.erase(beg, maps.end());
    cleanup();
}

template <typename T>
void Growable<T>::evict_all() {
    if (maps.empty())
        return;
    if (used) {
        auto dst = evicted_data.emplace_back(R{ new T[used], used });
        if (!dst.ptr)
            throw std::runtime_error{ std::format("new T[{}] failed ({} MiB)", used, used * sizeof(T) / 1048576.0) };
        C(hipMemcpy(dst.ptr, maps[0].ptr, used * sizeof(T), hipMemcpyDeviceToHost));
        evicted += used;
        used = 0;
    }
    for (auto rh : maps) {
        C(hipMemUnmap((hipDeviceptr_t)rh.ptr, rh.len * sizeof(T)));
        C(hipMemRelease(rh.h));
    }
    maps.clear();
    mapped = 0;
    for (auto vm : vmaps) {
        C(hipMemAddressFree((hipDeviceptr_t)vm.ptr, vm.len * sizeof(T)));
    }
    vmaps.clear();
    reserved = 0;
}

template <typename T>
void Growable<T>::remap(size_t new_max, bool force) {
    new_max = (new_max + chunk - 1) / chunk * chunk;
    if (!force && offset + new_max <= reserved)
        return;

    cleanup();

    hipDeviceptr_t new_ptr{};
    if (!force && !vmaps.empty()
            && hipMemAddressReserve(&new_ptr, (new_max - reserved) * sizeof(T), 
                alignof(T), (hipDeviceptr_t)(vmaps[0].ptr + reserved), 0) == hipSuccess
            && new_ptr == (hipDeviceptr_t)(vmaps[0].ptr + reserved)) {
        vmaps.emplace_back(R{ (T *)new_ptr, new_max - reserved }); 
        reserved = new_max;
        return;
    }
    if (new_ptr) { // remove accidentally created vmap
        C(hipMemAddressFree(new_ptr, (new_max - reserved) * sizeof(T)));
    }
    C(hipMemAddressReserve(&new_ptr, new_max * sizeof(T), alignof(T), 0, 0));
    offset = 0;
    auto o = (T *)new_ptr;
    for (auto &rh : maps) {
        C(hipMemUnmap((hipDeviceptr_t)rh.ptr, rh.len * sizeof(T)));
        rh.ptr = o;
        C(hipMemMap((hipDeviceptr_t)rh.ptr, rh.len * sizeof(T), 0, rh.h, 0));
        C(hipMemSetAccess((hipDeviceptr_t)rh.ptr, rh.len * sizeof(T), &adesc, 1));
        o += rh.len;
    }
    for (auto vm : vmaps)
        C(hipMemAddressFree((hipDeviceptr_t)vm.ptr, vm.len * sizeof(T)));
    vmaps.clear();
    vmaps.emplace_back(R{ (T *)new_ptr, new_max });
    reserved = new_max;
}

template <typename T>
void Growable<T>::cleanup() {
    while (!vmaps.empty() && vmaps.front().len <= offset) {
        auto vm = vmaps.front();
        C(hipMemAddressFree((hipDeviceptr_t)vm.ptr, vm.len * sizeof(T)));
        offset -= vm.len;
        reserved -= vm.len;
        vmaps.pop_front();
    }
    while (!vmaps.empty() && vmaps.back().len <= reserved - offset - mapped) {
        auto vm = vmaps.back();
        C(hipMemAddressFree((hipDeviceptr_t)vm.ptr, vm.len * sizeof(T)));
        reserved -= vm.len;
        vmaps.pop_back();
    }
}

std::string display(uint64_t byte) {
    if (byte < 1000ull)
        return std::format("{}", byte);
    if (byte < 1024 * 1024ull)
        return std::format("{:.2f} Ki", 1.0 * byte / 1024);
    if (byte < 1024 * 1024ull * 1024ull)
        return std::format("{:.2f} Mi", 1.0 * byte / 1024 / 1024);
    if (byte < 1024 * 1024ull * 1024ull * 1024ull)
        return std::format("{:.2f} Gi", 1.0 * byte / 1024 / 1024 / 1024);
    return std::format("{:.3f} TiB", 1.0 * byte / 1024 / 1024 / 1024 / 1024);
}

template <typename T>
void Growable<T>::mem_stat() const {
    std::cout << std::format(R"(
chunk:     {:10} = {}
reserved:  {:10} = {} ({} vmaps)
offset:    {:10} = {}
used:      {:10} = {}
mapped:    {:10} = {} ({} maps)
evicted:   {:10} = {} ({} pieces)
risk-free: {:10} = {}
)",
            display(chunk * sizeof(T)), chunk,
            display(reserved * sizeof(T)), reserved, vmaps.size(),
            display(offset * sizeof(T)), offset,
            display(used * sizeof(T)), used,
            display(mapped * sizeof(T)), mapped, maps.size(),
            display(evicted * sizeof(T)), evicted, evicted_data.size(),
            display(risk_free_size() * sizeof(T)), risk_free_size());
    for (auto vm : vmaps)
        std::cout << std::format("  vmaps[0x{:016x}:{:016x}) => {}\n",
                (ptrdiff_t)vm.ptr, (ptrdiff_t)(vm.ptr + vm.len), display(vm.len * sizeof(T)));
    for (auto rh : maps)
        std::cout << std::format("    maps[0x{:016x}:{:016x}) => {}\n",
                (ptrdiff_t)rh.ptr, (ptrdiff_t)(rh.ptr + rh.len), display(rh.len * sizeof(T)));
}

/*
int main() {
    Growable<float> gr{};
    std::string str;
    double sz;
    float *ptr{};
    while (true) {
        gr.mem_stat();
        if (ptr)
            std::cout << std::format("  ptr => 0x{:016x}\n", (uint64_t)ptr);
        std::cin >> str;
        if (str == "r") {
            std::cin >> sz;
            gr.remap((size_t)sz);
        } else if (str == "rf") {
            std::cin >> sz;
            gr.remap((size_t)sz, true);
        } else if (str == "cm" || str == "c") {
            std::cin >> sz;
            hiprandGenerator_t gen;
            C(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
            C(hiprandSetPseudoRandomGeneratorSeed(gen, (size_t)sz));
            C(hiprandGenerateUniform(gen, ptr, (size_t)sz));
            ptr += (size_t)sz;
            gr.commit((size_t)sz);
        } else if (str == "en" || str == "e") {
            std::cin >> sz;
            ptr = gr.get((size_t)sz);
        } else if (str == "x") {
            gr.compact();
        } else if (str == "e1") {
            gr.evict1();
        } else if (str == "ea") {
            gr.evict_all();
        } else if (str == "cl") {
            gr.cleanup();
        }
    }
}
*/
