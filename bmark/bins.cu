#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <algorithm>
#include <iostream>
#include <format>
#include <deque>
#include <ranges>
#include <vector>

#define C(ans) { chk_impl((ans), __FILE__, __LINE__); }

static inline void chk_impl(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        throw std::runtime_error{
            std::format("CUDA: {}: {} @ {}:{}\n",
                    hipGetErrorName(code), hipGetErrorString(code),
                    file, line) };
    }
}

static inline void chk_impl(hipError_t code, const char *file, int line) {
    const char *pn = "???", *ps = "???";
    hipDrvGetErrorName(code, &pn);
    hipDrvGetErrorString(code, &ps);
    if (code != hipSuccess) {
        throw std::runtime_error{
            std::format("CUDA Driver: {}: {} @ {}:{}\n", pn, ps, file, line) };
    }
}

static inline void chk_impl(hiprandStatus_t code, const char *file, int line) {
    if (code != HIPRAND_STATUS_SUCCESS) {
        throw std::runtime_error{
            std::format("hiprand: {} @ {}:{}\n", (int)code, file, line) };
    }
}

template <typename T>
class Growable {
    struct R {
        T *ptr;
        size_t len; // number of T
    };
    struct RH : R {
        hipMemGenericAllocationHandle_t h;
    };

    // in units of T
    std::deque<R> vmaps; // sum(vmaps, &R::len) == reserved
    size_t reserved; // offset + mapped <= reserved
    size_t offset;
    size_t used; // used <= mapped
    // vmaps[0].ptr + offset == maps[0].ptr
    std::deque<RH> maps; // sum(maps, &R::len) == mapped
    size_t mapped;
    std::vector<R> evicted_data;
    size_t evicted;
    size_t chunk; // granularity

    hipMemAllocationProp prop, uprop;
    hipMemAccessDesc adesc, uadesc;

public:
    explicit Growable(size_t max = 0);
    ~Growable();

    static_assert(std::is_trivially_constructible_v<T>, "T not trivially constructible");
    static_assert(std::is_trivially_copyable_v<T>, "T not trivially copyable");

    // return how many T can be written without crash
    [[nodiscard]] size_t risk_free_size() const { return mapped - used; }
    // re-organize all pa mappings s.t. reserved >= offset + new_reserved
    void remap(size_t new_max, bool force = false);
    // mark n of Ts are actually consumed
    void commit(size_t n) { used += n; }
    // free up unused pa
    void compact();
    // allocate a contiguous T[n]
    T *get(size_t n) {
        if (ensure(n))
            return vmaps[0].ptr + used;
        return nullptr;
    }

    void mem_stat() const;

    // make sure risk_free_size() >= n, and return the write-start point
    bool ensure(size_t n);

    // copy all useful data from the 0-th pa to evicted_data
    // does NOT free up pa
    void evict1();

    // copy all useful data to evicted_data
    // free up all pa
    void evict_all();

    // remove unused vmap
    void cleanup();
};

template <typename T>
Growable<T>::Growable(size_t max)
    : reserved{}, vmaps{}, offset{}, used{}, maps{},
      mapped{}, evicted_data{}, evicted{}, chunk{},
      prop{}, uprop{}, adesc{}, uadesc{} {

    int n; C(hipGetDeviceCount(&n)); // dark magic; don't touch

    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = 0; // TODO
    adesc.location = prop.location;
    adesc.flags = hipMemAccessFlagsProtReadWrite;

    uprop.type = hipMemAllocationTypePinned;
    uprop.location.type = hipMemLocationTypeHostNuma;
    uprop.location.id = 0; // TODO
    uadesc.location = uprop.location;
    uadesc.flags = hipMemAccessFlagsProtReadWrite;

    C(hipMemGetAllocationGranularity(&chunk, &prop, hipMemAllocationGranularityMinimum));
    chunk = (chunk + sizeof(T) - 1) / sizeof(T);
    remap(max);
};

template <typename T>
Growable<T>::~Growable() {
    for (auto rh : maps) {
        C(hipMemUnmap((hipDeviceptr_t)rh.ptr, rh.len * sizeof(T)));
        C(hipMemRelease(rh.h));
    }
    for (auto v : vmaps)
        C(hipMemAddressFree((hipDeviceptr_t)v.ptr, v.len * sizeof(T)));
}

template <typename T>
bool Growable<T>::ensure(size_t n) {
    if (mapped && used + n <= mapped) return true;

    auto sz = std::max((used + n - mapped + chunk - 1) / chunk, 1zu) * chunk;
    remap(mapped + sz); // vmap must not be empty

    hipMemGenericAllocationHandle_t h;
    hipError_t err;
    if ((err = hipMemCreate(&h, sz * sizeof(T), &prop, 0)) == hipSuccess) goto map;
    if (err != hipErrorOutOfMemory) C(err);
    if (!mapped) goto fail;
again:
    evict1();
    if (used + n <= mapped) return true;
    sz = std::max((used + n - mapped + chunk - 1) / chunk, 1zu) * chunk;
    if ((err = hipMemCreate(&h, sz * sizeof(T), &prop, 0)) == hipSuccess) goto map;
    if (used && err == hipErrorOutOfMemory) goto again;
    if (err != hipErrorOutOfMemory) C(err);
fail:
    evict_all();
    return false;
map:
    auto ptr = vmaps[0].ptr + offset + mapped;
    C(hipMemMap((hipDeviceptr_t)ptr, sz * sizeof(T), 0, h, 0));
    C(hipMemSetAccess((hipDeviceptr_t)ptr, sz * sizeof(T), &adesc, 1));
    maps.emplace_back(RH{ R{ ptr, sz }, h });
    mapped += sz;
    return true;
}

template <typename T>
void Growable<T>::evict1() {
    if (maps.empty())
        throw std::runtime_error{ "cannot evict: nothing was allocated" };

    auto src = maps.front();
    auto used1 = min(used, src.len);
    if (used1) {
        auto dst = evicted_data.emplace_back(R{ new T[used1], used1 });
        if (!dst.ptr)
            throw std::runtime_error{ std::format("new T[{}] failed ({} MiB)", used1, used1 * sizeof(T) / 1048576.0) };
        C(hipMemcpy(dst.ptr, src.ptr, used1 * sizeof(T), hipMemcpyDeviceToHost));
        evicted += used1;
    }
    if (used < src.len) {
        used = 0;
    } else {
        remap(mapped + src.len);
        src = maps.front();
        C(hipMemUnmap((hipDeviceptr_t)src.ptr, src.len * sizeof(T)));
        offset += src.len;
        C(hipMemMap((hipDeviceptr_t)src.ptr, src.len * sizeof(T), 0, src.h, 0));
        C(hipMemSetAccess((hipDeviceptr_t)src.ptr, src.len * sizeof(T), &adesc, 1));
        used -= src.len;
    }
}

template <typename T>
void Growable<T>::compact() {
    if (maps.empty()) {
        cleanup();
        return;
    }
    auto beg = std::ranges::lower_bound(maps, maps[0].ptr + used, std::less{}, &RH::ptr);
    for (auto it = beg; it != maps.end(); it++) {
        C(hipMemUnmap((hipDeviceptr_t)it->ptr, it->len * sizeof(T)));
        mapped -= it->len;
        C(hipMemRelease(it->h));
    }
    maps.erase(beg, maps.end());
    cleanup();
}

template <typename T>
void Growable<T>::evict_all() {
    if (maps.empty())
        return;
    if (used) {
        auto dst = evicted_data.emplace_back(R{ new T[used], used });
        if (!dst.ptr)
            throw std::runtime_error{ std::format("new T[{}] failed ({} MiB)", used, used * sizeof(T) / 1048576.0) };
        C(hipMemcpy(dst.ptr, maps[0].ptr, used * sizeof(T), hipMemcpyDeviceToHost));
        evicted += used;
        used = 0;
    }
    for (auto rh : maps) {
        C(hipMemUnmap((hipDeviceptr_t)rh.ptr, rh.len * sizeof(T)));
        C(hipMemRelease(rh.h));
    }
    maps.clear();
    mapped = 0;
    for (auto vm : vmaps) {
        C(hipMemAddressFree((hipDeviceptr_t)vm.ptr, vm.len * sizeof(T)));
    }
    vmaps.clear();
    reserved = 0;
}

template <typename T>
void Growable<T>::remap(size_t new_max, bool force) {
    new_max = (new_max + chunk - 1) / chunk * chunk;
    if (!force && offset + new_max <= reserved)
        return;

    cleanup();

    hipDeviceptr_t new_ptr{};
    if (!force && !vmaps.empty()
            && hipMemAddressReserve(&new_ptr, (new_max - reserved) * sizeof(T), 
                alignof(T), (hipDeviceptr_t)(vmaps[0].ptr + reserved), 0) == hipSuccess
            && new_ptr == (hipDeviceptr_t)(vmaps[0].ptr + reserved)) {
        vmaps.emplace_back(R{ (T *)new_ptr, new_max - reserved }); 
        reserved = new_max;
        return;
    }
    if (new_ptr) { // remove accidentally created vmap
        C(hipMemAddressFree(new_ptr, (new_max - reserved) * sizeof(T)));
    }
    C(hipMemAddressReserve(&new_ptr, new_max * sizeof(T), alignof(T), 0, 0));
    offset = 0;
    auto o = (T *)new_ptr;
    for (auto &rh : maps) {
        C(hipMemUnmap((hipDeviceptr_t)rh.ptr, rh.len * sizeof(T)));
        rh.ptr = o;
        C(hipMemMap((hipDeviceptr_t)rh.ptr, rh.len * sizeof(T), 0, rh.h, 0));
        C(hipMemSetAccess((hipDeviceptr_t)rh.ptr, rh.len * sizeof(T), &adesc, 1));
        o += rh.len;
    }
    for (auto vm : vmaps)
        C(hipMemAddressFree((hipDeviceptr_t)vm.ptr, vm.len * sizeof(T)));
    vmaps.clear();
    vmaps.emplace_back(R{ (T *)new_ptr, new_max });
    reserved = new_max;
}

template <typename T>
void Growable<T>::cleanup() {
    while (!vmaps.empty() && vmaps.front().len <= offset) {
        auto vm = vmaps.front();
        C(hipMemAddressFree((hipDeviceptr_t)vm.ptr, vm.len * sizeof(T)));
        offset -= vm.len;
        reserved -= vm.len;
        vmaps.pop_front();
    }
    while (!vmaps.empty() && vmaps.back().len <= reserved - offset - mapped) {
        auto vm = vmaps.back();
        C(hipMemAddressFree((hipDeviceptr_t)vm.ptr, vm.len * sizeof(T)));
        reserved -= vm.len;
        vmaps.pop_back();
    }
}

std::string display(uint64_t byte) {
    if (byte < 1000ull)
        return std::format("{}", byte);
    if (byte < 1024 * 1024ull)
        return std::format("{:.2f} Ki", 1.0 * byte / 1024);
    if (byte < 1024 * 1024ull * 1024ull)
        return std::format("{:.2f} Mi", 1.0 * byte / 1024 / 1024);
    if (byte < 1024 * 1024ull * 1024ull * 1024ull)
        return std::format("{:.2f} Gi", 1.0 * byte / 1024 / 1024 / 1024);
    return std::format("{:.3f} TiB", 1.0 * byte / 1024 / 1024 / 1024 / 1024);
}

template <typename T>
void Growable<T>::mem_stat() const {
    std::cout << std::format(R"(
chunk:     {:10} = {}
reserved:  {:10} = {} ({} vmaps)
offset:    {:10} = {}
used:      {:10} = {}
mapped:    {:10} = {} ({} maps)
evicted:   {:10} = {} ({} pieces)
risk-free: {:10} = {}
)",
            display(chunk * sizeof(T)), chunk,
            display(reserved * sizeof(T)), reserved, vmaps.size(),
            display(offset * sizeof(T)), offset,
            display(used * sizeof(T)), used,
            display(mapped * sizeof(T)), mapped, maps.size(),
            display(evicted * sizeof(T)), evicted, evicted_data.size(),
            display(risk_free_size() * sizeof(T)), risk_free_size());
    for (auto vm : vmaps)
        std::cout << std::format("  vmaps[0x{:016x}:{:016x}) => {}\n",
                (ptrdiff_t)vm.ptr, (ptrdiff_t)(vm.ptr + vm.len), display(vm.len * sizeof(T)));
    for (auto rh : maps)
        std::cout << std::format("    maps[0x{:016x}:{:016x}) => {}\n",
                (ptrdiff_t)rh.ptr, (ptrdiff_t)(rh.ptr + rh.len), display(rh.len * sizeof(T)));
}

int main() {
    Growable<float> gr{};
    std::string str;
    double sz;
    float *ptr{};
    while (true) {
        gr.mem_stat();
        if (ptr)
            std::cout << std::format("  ptr => 0x{:016x}\n", (uint64_t)ptr);
        std::cin >> str;
        if (str == "r") {
            std::cin >> sz;
            gr.remap((size_t)sz);
        } else if (str == "rf") {
            std::cin >> sz;
            gr.remap((size_t)sz, true);
        } else if (str == "cm" || str == "c") {
            std::cin >> sz;
            hiprandGenerator_t gen;
            C(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
            C(hiprandSetPseudoRandomGeneratorSeed(gen, (size_t)sz));
            C(hiprandGenerateUniform(gen, ptr, (size_t)sz));
            ptr += (size_t)sz;
            gr.commit((size_t)sz);
        } else if (str == "en" || str == "e") {
            std::cin >> sz;
            ptr = gr.get((size_t)sz);
        } else if (str == "x") {
            gr.compact();
        } else if (str == "e1") {
            gr.evict1();
        } else if (str == "ea") {
            gr.evict_all();
        } else if (str == "cl") {
            gr.cleanup();
        }
    }
}
