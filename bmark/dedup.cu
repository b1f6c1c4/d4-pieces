#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <stdexcept>
#include <format>
#include <iostream>
#include <hiprand.h>

#define C(ans) { chk_impl((ans), __FILE__, __LINE__); }

static inline void chk_impl(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        throw std::runtime_error{
            std::format("CUDA: {}: {} @ {}:{}\n",
                    hipGetErrorName(code), hipGetErrorString(code),
                    file, line) };
    }
}

static inline void chk_impl(hiprandStatus_t code, const char *file, int line) {
    if (code != HIPRAND_STATUS_SUCCESS) {
        throw std::runtime_error{
            std::format("hiprand: {} @ {}:{}\n", (int)code, file, line) };
    }
}

__device__ unsigned long long r_count, w_count;

__global__
void dedup_n2(
        uint64_t *data,
        unsigned long long n_data) {
    auto idx = threadIdx.x + (unsigned long long)blockIdx.x * blockDim.x;
    if (idx >= n_data)
        return;

    auto datum = data[idx];
    auto limit = __nv_atomic_load_n(&w_count, __NV_ATOMIC_ACQUIRE, __NV_THREAD_SCOPE_DEVICE);
    for (auto i = 0ull; i < limit; i++)
        if (data[i] == datum)
            return;

    auto out = __nv_atomic_fetch_add(&r_count, 1, __NV_ATOMIC_RELAXED, __NV_THREAD_SCOPE_DEVICE);
    data[out] = datum;
    __nv_atomic_fetch_add(&w_count, 1, __NV_ATOMIC_RELEASE, __NV_THREAD_SCOPE_DEVICE);
}

int main(int argc, char *argv[]) {
    auto sz = std::atoll(argv[1]);
    hiprandGenerator_t gen;
    C(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    C(hiprandSetPseudoRandomGeneratorSeed(gen, sz));
    uint64_t *ptr;
    C(hipMalloc(&ptr, sz * sizeof(uint64_t)));
    C(hiprandGenerateUniformDouble(gen, reinterpret_cast<double *>(ptr), sz));
    dedup_n2<<<sz / 512, 512>>>(ptr, sz);
}
