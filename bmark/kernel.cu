#include "hip/hip_runtime.h"
#include <vector>

#include <hiprand.h>

#include "../src/frow.h"
#include "../src/kernel.h"

#include <format>
#include <iostream>

#include "../src/naming.hpp"
#include "../src/known.hpp"

#include "../src/util.cuh"

static inline void chk_impl(hiprandStatus_t code, const char *file, int line) {
    if (code != HIPRAND_STATUS_SUCCESS) {
        throw std::runtime_error{
            std::format("hiprand: {} @ {}:{}\n", (int)code, file, line) };
    }
}

// defined in frow.cpp
extern std::optional<Naming> g_nme;
extern unsigned g_sym;

#define N_CHUNKS 47
__device__  RX                 ring_buffer[N_CHUNKS*CYC_CHUNK/sizeof(RX)];
__device__  unsigned long long n_outs;
__managed__ unsigned long long nrc, nwc;
__managed__ R *cfgs;

int main(int argc, char *argv[]) {
    if (argc != 9) {
        std::cout << std::format(
                "Usage: {} <min_m> <max_m> <min_n> <max_n> <board_n> <ea> <height> <n_cfgs>\n",
                argv[0]);
        return 1;
    }
    auto sym_C = ::getenv("C") && *::getenv("C");
    g_sym = sym_C ? 0b01101001u : 0b11111111u;
    auto min_m = std::atoi(argv[1]);
    auto max_m = std::atoi(argv[2]);
    auto min_n = std::atoi(argv[3]);
    auto max_n = std::atoi(argv[4]);
    auto board_n = std::atoi(argv[5]);
    auto ea = (uint8_t)std::strtol(argv[6], nullptr, 16);
    auto height = (unsigned)std::atoi(argv[7]);
    auto n_cfgs = (uint64_t)std::atoll(argv[8]);
    g_nme.emplace(
        (uint64_t)min_m, (uint64_t)max_m,
        (uint64_t)min_n, (uint64_t)max_n,
        board_n,
        sym_C ? known_C_shapes : known_shapes,
        sym_C ? shapes_C_count : shapes_count);

    compute_frow_on_cpu();

    auto szid = std::min(height - 1, 5u);
    auto fanoutL = h_frowInfoL[(ea >> 0) & 0xfu].sz[szid];
    auto fanoutR = h_frowInfoR[(ea >> 4) & 0xfu].sz[szid];

    prepare_kernels();

    hipStream_t stream;
    C(hipStreamCreate(&stream));

    frow_t *f0L, *f0R;
    std::cout << std::format("copy f0L({}), f0R({}) at szid={}\n", fanoutL, fanoutR, szid);
    C(hipMallocAsync(&f0L, fanoutL*sizeof(frow_t), stream));
    C(hipMallocAsync(&f0R, fanoutR*sizeof(frow_t), stream));
    C(hipMemcpyAsync(f0L, h_frowInfoL[(ea >> 0) & 0xfu].data,
                fanoutL*sizeof(frow_t), hipMemcpyHostToDevice, stream));
    C(hipMemcpyAsync(f0R, h_frowInfoR[(ea >> 0) & 0xfu].data,
                fanoutR*sizeof(frow_t), hipMemcpyHostToDevice, stream));

    std::cout << std::format("allocate {} cfgs\n", n_cfgs);
    C(hipMallocManaged(&cfgs, n_cfgs*sizeof(R)));
    std::cout << std::format("randomize {} cfgs\n", n_cfgs);
    hiprandGenerator_t gen;
    C(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    C(hiprandSetPseudoRandomGeneratorSeed(gen, 23336666));
    C(hiprandGenerateUniformDouble(gen,
                reinterpret_cast<double *>(cfgs), n_cfgs*sizeof(R)/sizeof(double)));

    auto pars = KSizing{ n_cfgs, fanoutL, fanoutR }.optimize();
    pars.erase(pars.begin() + 10, pars.end());
    for (auto res : pars) {
        unsigned long long tmp{};
        C(hipMemcpyToSymbolAsync(HIP_SYMBOL(n_outs), &tmp, 0,
                    sizeof(unsigned long long), hipMemcpyHostToDevice, stream));
        C(hipMemcpyToSymbolAsync(HIP_SYMBOL(nrc), &tmp, 0,
                    sizeof(unsigned long long), hipMemcpyHostToDevice, stream));
        C(hipMemcpyToSymbolAsync(HIP_SYMBOL(nwc), &tmp, 0,
                    sizeof(unsigned long long), hipMemcpyHostToDevice, stream));
        if (res.shmem_len) {
            std::cout << std::format("<<<{:9},{:5},{:5}B>>>[{}]/{:.02e} => ",
                    res.blocks, res.threads, res.shmem_len * sizeof(frow_t),
                    res.reverse ? "R" : "L",
                    res.fom());
        } else {
            std::cout << std::format("<<<{:9},{:5}>>>  [legacy]/{:.02e} => ",
                    res.blocks, res.threads,
                    res.fom());
        }
        std::cout.flush();

        KParamsFull kpf{ res, height,
            ring_buffer, &n_outs, N_CHUNKS,
            &nrc, &nwc, cfgs, ea, f0L, f0R };
        hipEvent_t start, stop;
        C(hipEventCreate(&start));
        C(hipEventCreate(&stop));
        C(hipEventRecord(start));
        kpf.launch(stream);
        C(hipPeekAtLastError());
        C(hipEventRecord(stop));
        C(hipEventSynchronize(stop));
        float ms;
        C(hipEventElapsedTime(&ms, start, stop));
        C(hipMemcpyFromSymbolAsync(&tmp, HIP_SYMBOL(n_outs), 0,
                    sizeof(unsigned long long), hipMemcpyDeviceToHost, stream));
        C(hipStreamSynchronize(stream));
        std::cout << std::format("{:.08f}ms, n_out={}\n", ms, n_outs);
        C(hipEventDestroy(start));
        C(hipEventDestroy(stop));
    }

    C(hipStreamDestroy(stream));
}
