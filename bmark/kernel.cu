#include "hip/hip_runtime.h"
#include <vector>

#include <hiprand.h>

#include <format>
#include <iostream>

#include "../src/frow.h"
#include "../src/kernel.h"
#include "../src/naming.hpp"
#include "../src/known.hpp"
#include "../src/record.cuh"
#include "../src/util.cuh"
#include "../src/sn.cuh"

static inline void chk_impl(hiprandStatus_t code, const char *file, int line) {
    if (code != HIPRAND_STATUS_SUCCESS) {
        throw std::runtime_error{
            std::format("hiprand: {} @ {}:{}\n", (int)code, file, line) };
    }
}

// defined in frow.cpp
extern std::optional<Naming> g_nme;
extern unsigned g_sym;

// #define N_CHUNKS 45
#define N_CHUNKS 9
__managed__ R *cfgs;

template <unsigned H>
__launch_bounds__(768, 2)
__global__ void fix_cfgs(unsigned long long n_cfgs) {
    auto idx = threadIdx.x + (uint64_t)blockDim.x * blockIdx.x;
    if (idx >= n_cfgs) return;
    auto cfg = parse_R<H>(cfgs[idx], 0x00);
    cfg.empty_area = ~0ull;
    switch (H) {
        case 8: cfg.nm_cnt = 0u; break;
        case 7: cfg.nm_cnt = (cfg.nm_cnt % 8u); break;
        case 6: cfg.nm_cnt = (cfg.nm_cnt % 14u); break;
        case 5: cfg.nm_cnt = (cfg.nm_cnt % 15u); break;
        default: cfg.nm_cnt = (cfg.nm_cnt % 16u); break;
    }
    d_sn(cfg.nm_cnt, cfg.ex);
    auto *nm = reinterpret_cast<uint8_t *>(cfg.ex);
    if (cfg.nm_cnt && nm[0] != 0xff) {
        auto ub = 0u; // [0,ub] are unique
        for (auto i = 1u; i < cfg.nm_cnt; i++) {
            if (nm[i] == 0xff)
                break;
            if (nm[i] != nm[ub])
                nm[++ub] = nm[i];
        }
        cfg.nm_cnt = ub;
    }
    for (auto i = cfg.nm_cnt; i < 16u; i++)
        nm[i] = 0xff;
    cfgs[idx] = assemble_R<H>(cfg);
}

void launch_fix_cfgs(unsigned H, unsigned long long n_cfgs, hipStream_t s) {
    switch (H) {
        case 7: fix_cfgs<7><<<(n_cfgs + 768 - 1) / 768, 768, 0, s>>>(n_cfgs); break;
        case 6: fix_cfgs<6><<<(n_cfgs + 768 - 1) / 768, 768, 0, s>>>(n_cfgs); break;
        case 5: fix_cfgs<5><<<(n_cfgs + 768 - 1) / 768, 768, 0, s>>>(n_cfgs); break;
        case 4: fix_cfgs<4><<<(n_cfgs + 768 - 1) / 768, 768, 0, s>>>(n_cfgs); break;
        case 3: fix_cfgs<3><<<(n_cfgs + 768 - 1) / 768, 768, 0, s>>>(n_cfgs); break;
        case 2: fix_cfgs<2><<<(n_cfgs + 768 - 1) / 768, 768, 0, s>>>(n_cfgs); break;
        case 1: fix_cfgs<1><<<(n_cfgs + 768 - 1) / 768, 768, 0, s>>>(n_cfgs); break;
    }
}

void show(const KParams &res) {
    if (res.shmem_len) {
        std::cout << std::format("<<<{:9},{:5},{:5}B>>>[{}]/{:.02e} => ",
                res.blocks, res.threads, res.shmem_len * sizeof(frow32_t),
                res.reverse ? "L" : "R",
                res.fom());
    } else {
        std::cout << std::format("<<<{:9},{:5}>>>  [legacy]/{:.02e} => ",
                res.blocks, res.threads,
                res.fom());
    }
}

int main(int argc, char *argv[]) {
    if (argc != 10) {
        std::cout << std::format(
                "Usage: {} <min_m> <max_m> <min_n> <max_n> <board_n> <ea> <height> <n_cfgs> <n_pars>\n",
                argv[0]);
        return 1;
    }
    auto sym_C = ::getenv("C") && *::getenv("C");
    g_sym = sym_C ? 0b01101001u : 0b11111111u;
    auto min_m = std::atoi(argv[1]);
    auto max_m = std::atoi(argv[2]);
    auto min_n = std::atoi(argv[3]);
    auto max_n = std::atoi(argv[4]);
    auto board_n = std::atoi(argv[5]);
    auto ea = (uint8_t)std::strtol(argv[6], nullptr, 16);
    auto height = (unsigned)std::atoi(argv[7]);
    auto n_cfgs = (uint64_t)std::atoll(argv[8]);
    auto n_pars = std::atoi(argv[9]);
    g_nme.emplace(
        (uint64_t)min_m, (uint64_t)max_m,
        (uint64_t)min_n, (uint64_t)max_n,
        board_n,
        sym_C ? known_C_shapes : known_shapes,
        sym_C ? shapes_C_count : shapes_count);

    show_gpu_devices();
    compute_frow_on_cpu();

    auto szid = std::min(height - 1, 5u);
    auto fanoutL = h_frowInfoL[(ea >> 0) & 0xfu].sz[szid];
    auto fanoutR = h_frowInfoR[(ea >> 4) & 0xfu].sz[szid];

    prepare_kernels();

    hipStream_t stream;
    C(hipStreamCreate(&stream));

    frow32_t *f0L, *f0R;
    std::cout << std::format("copy f0L({}), f0R({}) at szid={}\n", fanoutL, fanoutR, szid);
    C(hipMallocAsync(&f0L, fanoutL*sizeof(frow32_t), stream));
    C(hipMallocAsync(&f0R, fanoutR*sizeof(frow32_t), stream));
    C(hipMemcpyAsync(f0L, h_frowInfoL[(ea >> 0) & 0xfu].data32,
                fanoutL*sizeof(frow32_t), hipMemcpyHostToDevice, stream));
    C(hipMemcpyAsync(f0R, h_frowInfoR[(ea >> 0) & 0xfu].data32,
                fanoutR*sizeof(frow32_t), hipMemcpyHostToDevice, stream));

    std::cout << std::format("allocate {} cfgs\n", n_cfgs);
    C(hipMallocManaged(&cfgs, n_cfgs*sizeof(R)));
    std::cout << std::format("randomize {} cfgs\n", n_cfgs);
    hiprandGenerator_t gen;
    C(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    C(hiprandSetPseudoRandomGeneratorSeed(gen, 23336666));
    C(hiprandGenerateUniformDouble(gen,
                reinterpret_cast<double *>(cfgs), n_cfgs*sizeof(R)/sizeof(double)));
    std::cout << std::format("patch {} cfgs\n", n_cfgs);
    launch_fix_cfgs(height, n_cfgs, stream);
    C(hipPeekAtLastError());

    RX *ring_buffer;
    // C(hipMallocManaged(&ring_buffer, N_CHUNKS*CYC_CHUNK*sizeof(RX)));
    C(hipMallocAsync(&ring_buffer, N_CHUNKS*CYC_CHUNK*sizeof(RX), stream));
    unsigned long long *n_outs;
    C(hipMallocAsync(&n_outs, sizeof(unsigned long long), stream));

    auto pars = KSizing{ n_cfgs, fanoutL, fanoutR }.optimize();
    for (auto &res : pars)
        show(res), std::cout << "\n";
    if (pars.size() > n_pars)
        pars.erase(pars.begin() + n_pars, pars.end());
    // std::vector<KParams> pars;
    // pars.push_back(KParams{
    //         KSizing{ n_cfgs, fanoutL, fanoutR },
    //         false,
    //         84*2,
    //         768,
    //         50176 / sizeof(frow32_t)
    //         });
    // pars.push_back(KParams{
    //         KSizing{ n_cfgs, fanoutL, fanoutR },
    //         true,
    //         84*3,
    //         512,
    //         32768 / sizeof(frow32_t)
    //         });
    // pars.push_back(KParams{
    //         KSizing{ n_cfgs, fanoutL, fanoutR },
    //         false,
    //         (n_cfgs * fanoutL * fanoutR + 768 - 1) / 768,
    //         768,
    //         0,
    //         });
    for (auto &res : pars) {
        unsigned long long tmp{};
        C(hipMemcpyAsync(n_outs, &tmp,
                    sizeof(unsigned long long), hipMemcpyHostToDevice, stream));
        show(res);
        std::cout.flush();

        KParamsFull kpf{ res, height,
            ring_buffer, n_outs, N_CHUNKS,
            nullptr, nullptr, cfgs, ea, f0L, f0R };
        hipEvent_t start, stop;
        C(hipEventCreate(&start));
        C(hipEventCreate(&stop));
        C(hipEventRecord(start));
        kpf.launch(stream);
        C(hipPeekAtLastError());
        C(hipEventRecord(stop));
        C(hipEventSynchronize(stop));
        float ms;
        C(hipEventElapsedTime(&ms, start, stop));
        C(hipMemcpyAsync(&tmp, n_outs, sizeof(unsigned long long), hipMemcpyDeviceToHost, stream));
        C(hipStreamSynchronize(stream));
        std::cout << std::format("{} / {:.08f}ms\n", tmp, ms);
        C(hipEventDestroy(start));
        C(hipEventDestroy(stop));
    }

    C(hipStreamDestroy(stream));
}
