#include "hip/hip_runtime.h"
#include <vector>

#include <hiprand.h>
#include <hip/hip_runtime_api.h>
#include <cudaProfiler.h>
#include <cstdio>
#include <csignal>
#include <iomanip>
#include <unistd.h>
#include <readline/readline.h>
#include <readline/history.h>

#include <format>
#include <iostream>
#include <boost/algorithm/string/classification.hpp>
#include <boost/algorithm/string/split.hpp>

#include "../src/frow.h"
#include "../src/kernel.h"
#include "../src/naming.hpp"
#include "../src/known.hpp"
#include "../src/record.cuh"
#include "../src/util.cuh"
#include "../src/util.hpp"
#include "../src/sn.cuh"

// defined in frow.cpp
extern std::optional<Naming> g_nme;
extern unsigned g_sym;

// #define N_CHUNKS 45
#define N_CHUNKS 2

#define BLOCK 96

template <unsigned H>
__launch_bounds__(BLOCK, 1)
__global__ void fix_cfgs(R *cfgs, unsigned long long n_cfgs) {
    auto idx = threadIdx.x + (uint64_t)blockDim.x * blockIdx.x;
    if (idx >= n_cfgs) return;
    auto cfg = parse_R<H>(cfgs[idx], 0x00);
    cfg.empty_area = ~0ull;
    switch (H) {
        case 8: cfg.nm_cnt = 0u; break;
        case 7: cfg.nm_cnt = (cfg.nm_cnt % 8u); break;
        case 6: cfg.nm_cnt = (cfg.nm_cnt % 14u); break;
        case 5: cfg.nm_cnt = (cfg.nm_cnt % 15u); break;
        default: cfg.nm_cnt = (cfg.nm_cnt % 16u); break;
    }
    d_sn(cfg.nm_cnt, cfg.ex);
    if (cfg.nm_cnt && cfg.nm[0] != 0xff) {
        auto ub = 0u; // [0,ub] are unique
        for (auto i = 1u; i < cfg.nm_cnt; i++) {
            if (cfg.nm[i] == 0xff)
                break;
            if (cfg.nm[i] != cfg.nm[ub])
                cfg.nm[++ub] = cfg.nm[i];
        }
        cfg.nm_cnt = ub;
    } else {
        cfg.nm_cnt = 0;
    }
    for (auto i = cfg.nm_cnt; i < 16u; i++)
        cfg.nm[i] = 0xff;
    cfgs[idx] = assemble_R<H>(cfg);
}

void launch_fix_cfgs(unsigned H, R *cfgs, unsigned long long n_cfgs, hipStream_t s) {
    switch (H) {
        case 7: fix_cfgs<7><<<(n_cfgs + BLOCK - 1) / BLOCK, BLOCK, 0, s>>>(cfgs, n_cfgs); break;
        case 6: fix_cfgs<6><<<(n_cfgs + BLOCK - 1) / BLOCK, BLOCK, 0, s>>>(cfgs, n_cfgs); break;
        case 5: fix_cfgs<5><<<(n_cfgs + BLOCK - 1) / BLOCK, BLOCK, 0, s>>>(cfgs, n_cfgs); break;
        case 4: fix_cfgs<4><<<(n_cfgs + BLOCK - 1) / BLOCK, BLOCK, 0, s>>>(cfgs, n_cfgs); break;
        case 3: fix_cfgs<3><<<(n_cfgs + BLOCK - 1) / BLOCK, BLOCK, 0, s>>>(cfgs, n_cfgs); break;
        case 2: fix_cfgs<2><<<(n_cfgs + BLOCK - 1) / BLOCK, BLOCK, 0, s>>>(cfgs, n_cfgs); break;
        case 1: fix_cfgs<1><<<(n_cfgs + BLOCK - 1) / BLOCK, BLOCK, 0, s>>>(cfgs, n_cfgs); break;
    }
}

static KSizing ks;

int default_shmem(int threads) {
    if (threads <= 96)
        return 7168;
    else if (threads <= 128)
        return 11776;
    else if (threads <= 256)
        return 15872;
    else if (threads <= 384)
        return 24576;
    else if (threads <= 512)
        return 32768;
    else if (threads <= 768)
        return 50176;
    else
        return 101376;
}

static bool running = false;

using namespace std::string_literals;
// ChatGPT generated code, fixed 114514 bugs {{{
static const std::vector<std::string> THREADS{"32", "64", "96", "128", "192", "256", "384", "512", "768", "1024"};

// Readline completion function
char *command_generator(const char *text, int state) {
    static std::vector<std::string> matches;
    static size_t index;

    if (state == 0) {
        matches.clear();
        index = 0;

        std::vector<std::string> tokens;
        boost::split(tokens, rl_line_buffer, boost::is_any_of(" "), boost::token_compress_on);

        // Determine position in the input
        size_t pos = tokens.size();
        bool is_lr = !tokens.empty() && (tokens[0] == "L" || tokens[0] == "R");
        bool is_co = !tokens.empty() && (tokens[0] == "CL" || tokens[0] == "CR");
        bool is_legacy = !tokens.empty() && tokens[0] == "legacy";

        if (pos == 1) {
            if ("legacy"s.starts_with(text)) matches.push_back("legacy");
            if ("list"s.starts_with(text)) matches.push_back("list");
            if ("L"s.starts_with(text)) matches.push_back("L");
            if ("R"s.starts_with(text)) matches.push_back("R");
            if ("CL"s.starts_with(text)) matches.push_back("CL");
            if ("CR"s.starts_with(text)) matches.push_back("CR");
        } else if (pos == 2 && (is_lr || is_co || is_legacy)) {
            // Complete <threads>
            for (const std::string &thread : THREADS) {
                if (thread.starts_with(text))
                    matches.push_back(thread);
            }
        } else if (pos == 3 && is_lr) {
            // Complete <Ltile> (computed from `default_shmem`)
            auto threads = std::stoi(tokens[1]);
            auto Ltile = default_shmem(threads) / sizeof(frow32_t) / 2;
            matches.push_back(std::to_string(Ltile));
        } else if (pos == 4 && is_lr) {
            // Complete <Rtile> (computed from `default_shmem`)
            auto threads = std::stoi(tokens[1]);
            auto Rtile = default_shmem(threads) / sizeof(frow32_t) / 2;
            matches.push_back(std::to_string(Rtile));
        }
    }

    // Return next match
    if (index < matches.size()) {
        return strdup(matches[index++].c_str());
    }
    return nullptr;
}

// Readline wrapper function
char **custom_completer(const char *text, int , int ) {
    return rl_completion_matches(text, command_generator);
}

void handle_sigint(int sig) {
    if (running)
        exit(1);
    // Clear the current input line when Ctrl-C is pressed
    printf("\n");  // Move to a new line
    rl_on_new_line();  // Reset readline's internal state
    rl_replace_line("", 0);  // Clear the input buffer
    rl_redisplay();  // Refresh the prompt
}

// }}} ChatGPT generated code

struct FD {
    int fd;
    FD &operator<<(const char *str) {
        auto len = ::write(fd, str, std::strlen(str));
        if (errno == EINVAL || errno == EBADF)
            return *this;
        if (len != std::strlen(str))
            THROW("cannot write {} to fd {}: {}", str, fd, std::strerror(errno));
        return *this;
    }
    FD &operator<<(const std::string &str) {
        auto len = ::write(fd, str.c_str(), str.size());
        if (errno == EINVAL || errno == EBADF)
            return *this;
        if (len != str.size())
            THROW("cannot write {} to fd {}: {}", str, fd, std::strerror(errno));
        return *this;
    }
    FD &operator<<(std::integral auto v) {
        return *this << std::format("{}", v);
    }
    FD &operator<<(double v) {
        return *this << std::format("{:17}", v);
    }
    void flush() { }
};

int main(int argc, char *argv[]) {
    rl_attempted_completion_function = custom_completer;
    rl_variable_bind("show-all-if-ambiguous", "on");
    struct sigaction sa;
    sa.sa_handler = handle_sigint;
    sa.sa_flags = 0;
    sigemptyset(&sa.sa_mask);
    sigaction(SIGINT, &sa, NULL);

    if (argc != 9 && argc != 10) {
        std::print(
                "Usage: {} <min_m> <max_m> <min_n> <max_n> <board_n> <ea> <height> <n_cfgs> [<n_pars>]\n",
                argv[0]);
        return 1;
    }
    auto sym_C = ::getenv("C") && *::getenv("C");
    g_sym = sym_C ? 0b01101001u : 0b11111111u;
    auto min_m = std::atoi(argv[1]);
    auto max_m = std::atoi(argv[2]);
    auto min_n = std::atoi(argv[3]);
    auto max_n = std::atoi(argv[4]);
    auto board_n = std::atoi(argv[5]);
    auto ea = (uint8_t)std::strtol(argv[6], nullptr, 16);
    auto height = (unsigned)std::atoi(argv[7]);
    auto n_cfgs = (uint64_t)std::atoll(argv[8]);
    auto n_pars{ -1 };
    if (argc == 10)
        n_pars = std::atoi(argv[9]);
    g_nme.emplace(
            (uint64_t)min_m, (uint64_t)max_m,
            (uint64_t)min_n, (uint64_t)max_n,
            board_n,
            sym_C ? known_C_shapes : known_shapes,
            sym_C ? shapes_C_count : shapes_count);

    if (n_pars >= 0)
        show_gpu_devices();
    compute_frow_on_cpu(n_pars >= 0);

    auto szid = std::min(height - 1, 5u);
    auto fanoutL = h_frowInfoL[(ea >> 0) & 0xfu].sz[szid];
    auto fanoutR = h_frowInfoR[(ea >> 4) & 0xfu].sz[szid];

    prepare_kernels();

    hipStream_t stream;
    C(hipStreamCreate(&stream));

    // for unknown reason, accessing d_frowDataX gives invalid memory access
    // transfer_frow_to_gpu();
    frow_info_d f0L, f0R;
    std::print("copy f0L({}), f0R({}) at szid={}\n", fanoutL, fanoutR, szid);
    C(hipMallocAsync(&f0L.data32, fanoutL*sizeof(frow32_t), stream));
    C(hipMallocAsync(&f0R.data32, fanoutR*sizeof(frow32_t), stream));
    C(hipMemcpyAsync(f0L.data32, h_frowInfoL[(ea >> 0) & 0xfu].data32,
                fanoutL*sizeof(frow32_t), hipMemcpyHostToDevice, stream));
    C(hipMemcpyAsync(f0R.data32, h_frowInfoR[(ea >> 4) & 0xfu].data32,
                fanoutR*sizeof(frow32_t), hipMemcpyHostToDevice, stream));
    C(hipMallocAsync(&f0L.dataL, fanoutL*sizeof(uint32_t), stream));
    C(hipMallocAsync(&f0R.dataL, fanoutR*sizeof(uint32_t), stream));
    C(hipMemcpyAsync(f0L.dataL, h_frowInfoL[(ea >> 0) & 0xfu].dataL,
                fanoutL*sizeof(uint32_t), hipMemcpyHostToDevice, stream));
    C(hipMemcpyAsync(f0R.dataL, h_frowInfoR[(ea >> 4) & 0xfu].dataL,
                fanoutR*sizeof(uint32_t), hipMemcpyHostToDevice, stream));
    C(hipMallocAsync(&f0L.dataH, fanoutL*sizeof(uint32_t), stream));
    C(hipMallocAsync(&f0R.dataH, fanoutR*sizeof(uint32_t), stream));
    C(hipMemcpyAsync(f0L.dataH, h_frowInfoL[(ea >> 0) & 0xfu].dataH,
                fanoutL*sizeof(uint32_t), hipMemcpyHostToDevice, stream));
    C(hipMemcpyAsync(f0R.dataH, h_frowInfoR[(ea >> 4) & 0xfu].dataH,
                fanoutR*sizeof(uint32_t), hipMemcpyHostToDevice, stream));
    C(hipMallocAsync(&f0L.data0123, fanoutL*sizeof(uint32_t), stream));
    C(hipMallocAsync(&f0R.data0123, fanoutR*sizeof(uint32_t), stream));
    C(hipMemcpyAsync(f0L.data0123, h_frowInfoL[(ea >> 0) & 0xfu].data0123,
                fanoutL*sizeof(uint32_t), hipMemcpyHostToDevice, stream));
    C(hipMemcpyAsync(f0R.data0123, h_frowInfoR[(ea >> 4) & 0xfu].data0123,
                fanoutR*sizeof(uint32_t), hipMemcpyHostToDevice, stream));

    std::print("allocate {} cfgs\n", n_cfgs);
    R *cfgs;
    C(hipMalloc(&cfgs, n_cfgs*sizeof(R)));
    std::print("randomize {} cfgs\n", n_cfgs);
    hiprandGenerator_t gen;
    C(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    C(hiprandSetPseudoRandomGeneratorSeed(gen, 23336666));
    C(hiprandGenerate(gen, reinterpret_cast<unsigned *>(cfgs),
                n_cfgs*sizeof(R)/sizeof(unsigned)));
    std::print("sync\n");
    C(hipStreamSynchronize(stream));
    std::print("patch {} cfgs\n", n_cfgs);
    launch_fix_cfgs(height, cfgs, n_cfgs, stream);
    C(hipPeekAtLastError());

    RX *ring_buffer;
    // C(hipMallocManaged(&ring_buffer, N_CHUNKS*CYC_CHUNK*sizeof(RX)));
    C(hipMallocAsync(&ring_buffer, N_CHUNKS*CYC_CHUNK*sizeof(RX), stream));
    unsigned long long *n_outs;
    C(hipMallocAsync(&n_outs, sizeof(unsigned long long), stream));
    unsigned long long *perf;
    C(hipMallocAsync(&perf, 4 * sizeof(long long), stream));
    C(hipStreamSynchronize(stream));

    hipDeviceProp_t prop;
    C(hipGetDeviceProperties(&prop, 0));

    FD csv{ 3 };
    csv << "n_cfgs,f0Lsz,f0Rsz,reverse,blocks,threads,Ltile,Rtile,ms,fom,height,ea,n_outs,clockRate,oc,e,perf_lr,perf_n,perf_tile,compI,ex\n";

    auto launch = [&](const KParams &kp) {
        running = true;
        C(hipMemsetAsync(n_outs, 0, sizeof(unsigned long long), stream));
        C(hipMemsetAsync(perf, 0, 4 * sizeof(unsigned long long), stream));
        std::cout << kp.to_string(false) << " => ";
        std::cout.flush();
        std::cout.flush();

        KParamsFull kpf{ kp, height,
            ring_buffer, n_outs, N_CHUNKS,
            nullptr, nullptr, cfgs, ea,
            f0L, // d_frowDataL[0][ea >> 0 & 0xfu],
            f0R, // d_frowDataR[0][ea >> 4 & 0xfu],
            perf };
        hipEvent_t start, stop;
        C(hipEventCreate(&start));
        C(hipEventCreate(&stop));
        C(hipEventRecord(start, stream));
        kpf.launch(stream);
        C(hipPeekAtLastError());
        C(hipEventRecord(stop, stream));
        C(hipEventSynchronize(stop));
        float ms;
        C(hipEventElapsedTime(&ms, start, stop));
        unsigned long long outs;
        C(hipMemcpyAsync(&outs, n_outs, sizeof(unsigned long long), hipMemcpyDeviceToHost, stream));
        unsigned long long perfs[4];
        C(hipMemcpyAsync(&perfs, perf, 4 * sizeof(unsigned long long), hipMemcpyDeviceToHost, stream));
        C(hipStreamSynchronize(stream));
        auto oc = std::min(16u, 1536u / kpf.threads) * 84; // max blocks per device
        auto e = ((kpf.blocks() + oc - 1) / oc);
        // auto tpg = kpb.blocks * kpb.threads;
        // auto iterations = (kpf.n_cfgs + tpg - 1) / tpg;
        auto rt = (kpf.n_cfgs + kpf.threads - 1) / kpf.threads * kpf.threads
            * 1000.0 * prop.clockRate / e;
        auto perf_lr = perfs[0];
        auto perf_n = perfs[1];
        auto perf_tile = perfs[2];
        auto perf_comp = perfs[3];
        auto ex = 1000 * (perf_lr + perf_n + perf_tile) / rt / ms;
        std::print("{} / {:.01f}ms = ({:>7}+{:>7}+{:>7})*{} I{:.02f}% E{:.02f}% raw({:>7}+{:>7}+{:>7})\n",
                outs, ms,
                display(perf_lr / rt / ex / e), display(perf_n / rt / ex / e), display(perf_tile / rt / ex / e),
                e,
                100.0 * perf_comp / perf_tile, 100.0 * ex,
                display(perf_lr / rt), display(perf_n / rt), display(perf_tile / rt));
        running = false;
        C(hipEventDestroy(start));
        C(hipEventDestroy(stop));
        csv << kpf.n_cfgs << ",";
        csv << kpf.f0Lsz << ",";
        csv << kpf.f0Rsz << ",";
        switch (kpf.ty) {
            case KKind::Legacy: csv << "legacy"; break;
            case KKind::CoalescedR: csv << "CR"; break;
            case KKind::CoalescedL: csv << "CL"; break;
            case KKind::TiledStandard: csv << "R"; break;
            case KKind::TiledReversed: csv << "L"; break;
        }
        csv << ",";
        csv << kpf.blocks() << ",";
        csv << kpf.threads << ",";
        csv << kpf.Ltile << ",";
        csv << kpf.Rtile << ",";
        csv << ms << ",";
        csv << kpf.fom() << ",";
        csv << kpf.height << ",";
        csv << (int)kpf.ea << ",";
        csv << outs << ",";
        csv << prop.clockRate << ",";
        csv << oc << ",";
        csv << e << ",";
        csv << 1e6 * perf_lr / rt / ex / e << ",";
        csv << 1e6 * perf_n / rt / ex / e << ",";
        csv << perf_tile / rt / ex / e << ",";
        csv << 1.0 * perf_comp / perf_tile << ",";
        csv << ex << "\n";
    };

    ks = KSizing{ n_cfgs, fanoutL, fanoutR };
    KParams kp;
    std::cout << R"(<COMMAND> ::=)" << "\n";
    std::cout << R"(    | "list")" << "\n";
    std::cout << R"(    | "legacy"    <threads> [<n_cfg>])" << "\n";
    std::cout << R"(    | ("CL"|"CR") <threads> [<n_cfg>])" << "\n";
    std::cout << R"(    | ("L"|"R")   <threads> [<Ltile> <Rtile> [<n_cfg>]])" << "\n";
    while (n_pars == -1) {
        auto input = readline("> ");
        if (input == nullptr)
            break;

        std::vector<std::string> tokens;
        std::string line{ input };
        free(input);
        boost::split(tokens, line, boost::is_any_of(" "), boost::token_compress_on);
        if (!tokens.empty() && tokens.back().empty())
            tokens.pop_back();
        if (tokens.empty())
            continue;
        if (tokens[0] == "list") {
            auto pars = ks.optimize();
            for (auto i = 0zu; i < pars.size() && i < 20zu; i++)
                pars[i].fom(true);
            continue;
        }
        if (tokens.size() < 2 || tokens.size() > 5) {
            std::cout << "invalid <KParams>\n";
            continue;
        }
        if (tokens[0] == "legacy") {
            kp = KParams{ ks, KKind::Legacy };
            kp.threads = std::stoull(tokens[1]);
            if (tokens.size() >= 4)
                kp.n_cfgs = std::stoull(tokens[3]);
        } else if (tokens[0] == "CL" || tokens[0] == "CR") {
            kp = KParams{ ks, KKind::CoalescedR };
            if (tokens[0] == "CL")
                kp.ty = KKind::CoalescedR;
            kp.threads = std::stoull(tokens[1]);
            if (tokens.size() >= 4)
                kp.n_cfgs = std::stoull(tokens[3]);
        } else if (tokens[0] == "L" || tokens[0] == "R") {
            kp = KParams{ ks, KKind::TiledStandard };
            if (tokens[0] == "L")
                kp.ty = KKind::TiledReversed;
            kp.threads = std::stoull(tokens[1]);
            if (tokens.size() >= 5)
                kp.n_cfgs = std::stoull(tokens[4]);
            if (tokens.size() < 3) {
                kp.Ltile = default_shmem(kp.threads) / sizeof(frow32_t) / 2;
                kp.Rtile = default_shmem(kp.threads) / sizeof(frow32_t) / 2;
            } else {
                kp.Ltile = std::stoll(tokens[2]);
                kp.Rtile = std::stoll(tokens[3]);
            }
        } else {
            continue;
        }
        add_history(line.c_str());
        kp.fom(true);
        launch(kp);
    }

    if (n_pars >= 0) {
        C(hipProfilerStart());
        auto pars = ks.optimize();
        if (pars.size() > n_pars)
            pars.erase(pars.begin() + n_pars, pars.end());
        for (auto it = pars.rbegin(); it != pars.rend(); it++)
            it->fom(true);
        for (auto &res : pars) {
            launch(res);
        }
    }

    C(hipStreamDestroy(stream));
    C(hipProfilerStop());
}
